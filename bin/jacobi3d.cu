#include "hip/hip_runtime.h"
#include <cmath>

#include <nvToolsExt.h>

#include "argparse/argparse.hpp"

#include "stencil/stencil.hpp"

#include "statistics.hpp"

const float COLD_TEMP = 0;
const float HOT_TEMP = 1;

/*! set compute region to zero
 */
/* Apply the stencil to the coordinates in `reg`
 */
__global__ void init_kernel(Accessor<float> dst, const Rect3 reg, const Rect3 cReg //<! the entire compute region
) {

  for (int64_t z = reg.lo.z + blockIdx.z * blockDim.z + threadIdx.z; z < reg.hi.z; z += gridDim.z * blockDim.z) {
    for (int64_t y = reg.lo.y + blockIdx.y * blockDim.y + threadIdx.y; y < reg.hi.y; y += gridDim.y * blockDim.y) {
      for (int64_t x = reg.lo.x + blockIdx.x * blockDim.x + threadIdx.x; x < reg.hi.x; x += gridDim.x * blockDim.x) {
        Dim3 o(x, y, z);
        dst[o] = (HOT_TEMP + COLD_TEMP) / 2;
      }
    }
  }
}

__device__ int64_t dist(const Dim3 a, const Dim3 b) {
  return __fsqrt_rn(float((a.x - b.x) * (a.x - b.x) + (a.y - b.y) * (a.y - b.y) + (a.z - b.z) * (a.z - b.z)));
}

/* Apply a 3d jacobi stencil to `reg`

   Since the library only supports periodic boundary conditions right now,
   fix part of the middle of the compute region at 1 and part at 0
 */
__global__ void stencil_kernel(Accessor<float> dst, const Accessor<float> src,
                               const Rect3 myReg, //<! the region i should modify
                               const Rect3 cReg   //<! the entire compute region
) {

  // x = 1/3, y = 1/2, z = 1/2
  const Dim3 hotCenter(cReg.lo.x + (cReg.hi.x - cReg.lo.x) / 3, (cReg.lo.y + cReg.hi.y) / 2,
                       (cReg.lo.z + cReg.hi.z) / 2);
  const Dim3 coldCenter(cReg.lo.x + (cReg.hi.x - cReg.lo.x) * 2 / 3, (cReg.lo.y + cReg.hi.y) / 2,
                        (cReg.lo.z + cReg.hi.z) / 2);
  const int sphereRadius = (cReg.hi.x - cReg.lo.x) / 10;

  for (int z = myReg.lo.z + blockIdx.z * blockDim.z + threadIdx.z; z < myReg.hi.z; z += gridDim.z * blockDim.z) {
    for (int y = myReg.lo.y + blockIdx.y * blockDim.y + threadIdx.y; y < myReg.hi.y; y += gridDim.y * blockDim.y) {
      for (int x = myReg.lo.x + blockIdx.x * blockDim.x + threadIdx.x; x < myReg.hi.x; x += gridDim.x * blockDim.x) {
        Dim3 o(x, y, z);

        /* a sphere 1/10 of the CR in radius and x = 1/3 of the way over is set hot
           a similar sphere of cold is at x = 2/3
        */
        if (dist(o, hotCenter) <= sphereRadius) {
          dst[o] = HOT_TEMP;
        } else if (dist(o, coldCenter) <= sphereRadius) {
          dst[o] = COLD_TEMP;
        } else {
          float px = src[o + Dim3(1, 0, 0)];
          float mx = src[o + Dim3(-1, 0, 0)];
          float py = src[o + Dim3(0, 1, 0)];
          float my = src[o + Dim3(0, -1, 0)];
          float pz = src[o + Dim3(0, 0, 1)];
          float mz = src[o + Dim3(0, 0, -1)];

          float val = 0;
          val += px;
          val += mx;
          val += py;
          val += my;
          val += pz;
          val += mz;
          val /= 6;
          dst[o] = val;
        }
      }
    }
  }
}

int main(int argc, char **argv) {

  bool useStaged = false;
  bool useCudaAwareMPI = false;
  bool useColo = false;
  bool useMemcpyPeer = false;
  bool useKernel = false;

  bool trivial = false;
  bool noOverlap = false;
  bool paraview = false;

  size_t x = 512;
  size_t y = 512;
  size_t z = 512;

  std::string prefix;

  int iters;
  int checkpointPeriod = -1;

  argparse::Parser parser("a cwpearson/argparse-powered CLI app");
  // clang-format off
  parser.add_flag(useStaged, "--staged")->help("Enable RemoteSender/Recver");
#if STENCIL_USE_CUDA_AWARE_MPI == 1
  parser.add_flag(useCudaAwareMPI, "--cuda-aware-mpi"->help("Enable CudaAwareMpiSender/Recver");
#endif
  parser.add_flag(useColo, "--colo")->help("Enable ColocatedHaloSender/Recver");
  parser.add_flag(useMemcpyPeer, "--peer")->help("Enable PeerAccessSender");
  parser.add_flag(useKernel, "--kernel")->help("Enable PeerCopySender");
  parser.add_flag(trivial, "--trivial")->help("Skip node-aware placement");
  parser.add_flag(noOverlap, "--no-overlap")->help("Don't overlap communication and computation");
  parser.add_option(prefix, "--prefix")->help("prefix for paraview files");
  parser.add_flag(paraview, "--paraview")->help("dump paraview files");
  parser.add_option(iters, "--iters", "-n")->help("number of iterations");
  parser.add_option(checkpointPeriod, "--period", "-q")->help("iterations between checkpoints");
  parser.add_positional(x)->required();
  parser.add_positional(y)->required();
  parser.add_positional(z)->required();
  // clang-format on

  if (!parser.parse(argc, argv)) {
    std::cerr << parser.help() << "\n";
    exit(EXIT_FAILURE);
  }

if (parser.need_help()) {
    std::cerr << parser.help() << "\n";
    return 0;
}

  // default checkpoint 10 times
  if (checkpointPeriod <= 0) {
    checkpointPeriod = iters / 10;
  }

  MPI_Init(&argc, &argv);

  int size;
  int rank;
  MPI_Comm_size(MPI_COMM_WORLD, &size);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);

  int devCount;
  CUDA_RUNTIME(hipGetDeviceCount(&devCount));

  // only works if no GPUs are overloaded
  int numSubdoms;
  {
    MpiTopology topo(MPI_COMM_WORLD);
    numSubdoms = size / topo.colocated_size() * devCount;
  }

  if (0 == rank) {
    std::cerr << "assuming " << numSubdoms << " subdomains\n";
  }


  /* scaling the cube with the number of GPUs caused wierd behavior
     for certain sizes due to the partitioner.
      Now, we'll just grow the domain by the reverse of the partitioning algorithm to keep each GPU
      having the same aspect ratio, to keep the performance more understandable

      The parition algorithm takes the pfs from largest to smallest, and recursively divides the longest axis
      so, we take the pfs smallest to largest and scale the smallest axis up
  */

  {
    std::vector<int64_t> pfs = prime_factors(numSubdoms);
    for (int i = pfs.size() - 1; i >= 0; --i) {
      if (x < y && x < z) {
        x *= pfs[i];
      } else if (y < z) {
        y *= pfs[i];
      } else {
        z *= pfs[i];
      }
    }
  }

  hipDeviceProp_t prop;
  CUDA_RUNTIME(hipGetDeviceProperties(&prop, 0));

  MethodFlags methods = MethodFlags::None;
  if (useStaged) {
    methods |= MethodFlags::CudaMpi;
  }
  if (useCudaAwareMPI) {
    methods |= MethodFlags::CudaAwareMpi;
  }
  if (useColo) {
    methods |= MethodFlags::CudaMpiColocated;
  }
  if (useMemcpyPeer) {
    methods |= MethodFlags::CudaMemcpyPeer;
  }
  if (useKernel) {
    methods |= MethodFlags::CudaKernel;
  }
  if (MethodFlags::None == methods) {
    methods = MethodFlags::All;
  }

  PlacementStrategy strategy = PlacementStrategy::NodeAware;
  if (trivial) {
    strategy = PlacementStrategy::Trivial;
  }

  bool overlap = true;
  if (noOverlap) {
    overlap = false;
  }

  Radius radius = Radius::constant(0);
  // x
  radius.dir(1, 0, 0) = 1;
  radius.dir(-1, 0, 0) = 1;
  // y
  radius.dir(0, 1, 0) = 1;
  radius.dir(0, -1, 0) = 1;
  // z
  radius.dir(0, 0, 1) = 1;
  radius.dir(0, 0, -1) = 1;
  // radius.set_face(1);


  Statistics iterTime;

  {
    DistributedDomain dd(x, y, z);

    dd.set_methods(methods);
    dd.set_radius(radius);
    dd.set_placement(strategy);

    auto dh = dd.add_data<float>("d");

    dd.realize();

    MPI_Barrier(MPI_COMM_WORLD);

    Rect3 computeRegion = dd.get_compute_region();

    // create a compute stream for each local domain
    std::vector<RcStream> computeStreams(dd.domains().size());
    for (size_t di = 0; di < dd.domains().size(); ++di) {
      computeStreams[di] = RcStream(dd.domains()[di].gpu());
    }

    // init current values
    std::cerr << "init\n";
    for (size_t di = 0; di < dd.domains().size(); ++di) {
      auto &d = dd.domains()[di];
      Rect3 reg = d.get_compute_region();
      const Accessor<float> src = d.get_curr_accessor<float>(dh);
      dim3 dimBlock = Dim3::make_block_dim(reg.extent(), 512);
      dim3 dimGrid = (reg.extent() + Dim3(dimBlock) - 1) / Dim3(dimBlock);
      d.set_device();
      init_kernel<<<dimGrid, dimBlock, 0, computeStreams[di]>>>(src, reg, computeRegion);
    }

    // wait for init to complete
    for (auto &s : computeStreams) {
      CUDA_RUNTIME(hipStreamSynchronize(s));
    }

    if (paraview) {
      dd.write_paraview(prefix + "jacobi3d_init");
    }

    const std::vector<Rect3> interiors = dd.get_interior();
    const std::vector<std::vector<Rect3>> exteriors = dd.get_exterior();

    for (int iter = 0; iter < iters; ++iter) {

      double elapsed = MPI_Wtime();

      if (overlap) {
        // launch operations on interior, safe to compute on before exchange
        for (size_t di = 0; di < dd.domains().size(); ++di) {
          auto &d = dd.domains()[di];
          const Rect3 mr = interiors[di];
          const Accessor<float> src0 = d.get_curr_accessor<float>(dh);
          const Accessor<float> dst0 = d.get_next_accessor<float>(dh);
          nvtxRangePush("launch");
          // if (0 == rank)
          //   std::cerr << rank << ": launch on region=" << mr << " (interior)\n";
          dim3 dimBlock = Dim3::make_block_dim(mr.extent(), 256);
          dim3 dimGrid = (mr.extent() + Dim3(dimBlock) - 1) / Dim3(dimBlock);
          d.set_device();
          stencil_kernel<<<dimGrid, dimBlock, 0, computeStreams[di]>>>(dst0, src0, mr, computeRegion);
          CUDA_RUNTIME(hipGetLastError());
          nvtxRangePop(); // launch
        }
      }

      // exchange halos: update ghost elements with current values from neighbors
      // if (0 == rank)
      //   std::cerr << rank << ": exchange\n";
      dd.exchange();

      if (overlap) {
        // operate on exterior now that ghost values are right
        for (size_t di = 0; di < dd.domains().size(); ++di) {
          auto &d = dd.domains()[di];
          const Accessor<float> src = d.get_curr_accessor<float>(dh);
          const Accessor<float> dst = d.get_next_accessor<float>(dh);
          for (size_t si = 0; si < exteriors[di].size(); ++si) {
            nvtxRangePush("launch");
            const Rect3 mr = exteriors[di][si];
            // if (0 == rank)
            //   std::cerr << rank << ": launch on region=" << mr << " (exterior)\n";
            dim3 dimBlock = Dim3::make_block_dim(mr.extent(), 256);
            dim3 dimGrid = (mr.extent() + Dim3(dimBlock) - 1) / Dim3(dimBlock);
            d.set_device();
            stencil_kernel<<<dimGrid, dimBlock, 0, computeStreams[di]>>>(dst, src, mr, computeRegion);
            CUDA_RUNTIME(hipGetLastError());
            nvtxRangePop(); // launch
          }
        }
      } else {
        // launch operations on compute region now that ghost values are right
        for (size_t di = 0; di < dd.domains().size(); ++di) {
          auto &d = dd.domains()[di];
          const Rect3 mr = d.get_compute_region();
          const Accessor<float> src = d.get_curr_accessor<float>(dh);
          const Accessor<float> dst = d.get_next_accessor<float>(dh);
          nvtxRangePush("launch (whole)");
          // if (0 == rank)
          // std::cerr << rank << ": launch on region=" << mr << " (whole)\n";
          d.set_device();
          dim3 dimBlock = Dim3::make_block_dim(mr.extent(), 256);
          dim3 dimGrid = (mr.extent() + Dim3(dimBlock) - 1) / Dim3(dimBlock);
          stencil_kernel<<<dimGrid, dimBlock, 0, computeStreams[di]>>>(dst, src, mr, computeRegion);
          CUDA_RUNTIME(hipGetLastError());
          nvtxRangePop(); // launch (whole)
        }
      }

      // wait for stencil to complete before swapping pointers
      for (auto &s : computeStreams) {
        CUDA_RUNTIME(hipStreamSynchronize(s));
      }

      // current = next
      dd.swap();

      elapsed = MPI_Wtime() - elapsed;
      MPI_Allreduce(MPI_IN_PLACE, &elapsed, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
      iterTime.insert(elapsed);

      if (paraview && (iter % checkpointPeriod == 0)) {
        dd.write_paraview(prefix + "jacobi3d_" + std::to_string(iter));
      }
    }

    if (paraview) {
      dd.write_paraview(prefix + "jacobi3d_final");
    }

  } // send domains out of scope before MPI_Finalize


  if (0 == mpi::world_rank()) {
    std::string methodStr;
    if (methods && MethodFlags::CudaMpi) {
      methodStr += methodStr.empty() ? "" : ",";
      methodStr += "staged";
    }
    if (methods && MethodFlags::CudaAwareMpi) {
      methodStr += methodStr.empty() ? "" : "/";
      methodStr += "cuda-aware";
    }
    if (methods && MethodFlags::CudaMpiColocated) {
      methodStr += methodStr.empty() ? "" : "/";
      methodStr += "colo";
    }
    if (methods && MethodFlags::CudaMemcpyPeer) {
      methodStr += methodStr.empty() ? "" : "/";
      methodStr += "peer";
    }
    if (methods && MethodFlags::CudaKernel) {
      methodStr += methodStr.empty() ? "" : "/";
      methodStr += "kernel";
    }

    std::cout << "jacobi3d," << methodStr << "," << size << "," << devCount << "," << x << "," << y << "," << z << ","
              << iterTime.min() << "," << iterTime.trimean() << "\n";
  }

  MPI_Finalize();

  return 0;
}
