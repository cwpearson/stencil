#include "hip/hip_runtime.h"
#include <chrono>
#include <cmath>
#include <thread>

#include <nvToolsExt.h>

#include <cxxopts/cxxopts.hpp>

#include "stencil/stencil.hpp"

/*! set compute region to dst[x,y,z] = sin(x+y+z + origin.x + origin.y + origin.z)
 */
template <typename T>
__global__ void init_kernel(Accessor<T> dst,    //<! [out] pointer to beginning of allocation
                            const Rect3 cr,     //<! [in] compute region
                            const double period //<! [in] sine wave period
) {
  for (int64_t z = cr.lo.z + blockIdx.z * blockDim.z + threadIdx.z; z < cr.hi.z; z += gridDim.z * blockDim.z) {
    for (int64_t y = cr.lo.y + blockIdx.y * blockDim.y + threadIdx.y; y < cr.hi.y; y += gridDim.y * blockDim.y) {
      for (int64_t x = cr.lo.x + blockIdx.x * blockDim.x + threadIdx.x; x < cr.hi.x; x += gridDim.x * blockDim.x) {
        dst[Dim3(x, y, z)] = sin(2 * 3.14159 / period * x + 2 * 3.14159 / period * y + 2 * 3.14159 / period * z);
      }
    }
  }
}

/* Apply the stencil to the coordinates in `reg`
 */
__global__ void stencil_kernel(Accessor<float> dst, const Accessor<float> src, const Rect3 reg) {

  for (int64_t z = reg.lo.z + blockIdx.z * blockDim.z + threadIdx.z; z < reg.hi.z; z += gridDim.z * blockDim.z) {
    for (int64_t y = reg.lo.y + blockIdx.y * blockDim.y + threadIdx.y; y < reg.hi.y; y += gridDim.y * blockDim.y) {
      for (int64_t x = reg.lo.x + blockIdx.x * blockDim.x + threadIdx.x; x < reg.hi.x; x += gridDim.x * blockDim.x) {
        Dim3 o(x, y, z);
        float val = 0;
        val += src[o + Dim3(-1, 0, 0)];
        val += src[o + Dim3(0, -1, 0)];
        val += src[o + Dim3(0, 0, -1)];
        val += src[o + Dim3(1, 0, 0)];
        val += src[o + Dim3(0, 1, 0)];
        val += src[o + Dim3(0, 0, 1)];
        val /= 6;
        dst[o] = val;
      }
    }
  }
}

int main(int argc, char **argv) {

  cxxopts::Options options("MyProgram", "One line description of MyProgram");
  // clang-format off
  options.add_options()
  ("h,help", "Show help")
  ("remote", "Enable RemoteSender/Recver")
  ("colocated", "Enable ColocatedHaloSender/Recver")
  ("peer", "Enable PeerAccessSender")
  ("kernel", "Enable PeerCopySender")
  ("trivial", "Skip node-aware placement")
  ("x", "x dim", cxxopts::value<int>()->default_value("512"))
  ("y", "y dim", cxxopts::value<int>()->default_value("512"))
  ("z", "z dim", cxxopts::value<int>()->default_value("512"))
  ("f,file", "File name", cxxopts::value<std::string>());
  // clang-format on

  auto result = options.parse(argc, argv);

  if (result["help"].as<bool>()) {
    std::cerr << options.help();
    exit(EXIT_SUCCESS);
  }

  MPI_Init(&argc, &argv);

  int size;
  int rank;
  MPI_Comm_size(MPI_COMM_WORLD, &size);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);

  int devCount;
  CUDA_RUNTIME(hipGetDeviceCount(&devCount));

  int numSubdoms;
  {
    MpiTopology topo(MPI_COMM_WORLD);
    numSubdoms = size / topo.colocated_size() * devCount;
  }

  if (0 == rank) {
    std::cout << "assuming " << numSubdoms << " subdomains\n";
  }

  double kernelMillis = 50;
  size_t x = result["x"].as<int>();
  size_t y = result["y"].as<int>();
  size_t z = result["z"].as<int>();

  hipDeviceProp_t prop;
  CUDA_RUNTIME(hipGetDeviceProperties(&prop, 0));
  if (std::string("Tesla V100-SXM2-32GB") == prop.name) {
    kernelMillis = 20.1;
  } else if (std::string("Tesla P100-SXM2-16GB") == prop.name) {
    kernelMillis = 34.1;
  } else {
    if (0 == rank) {
      std::cerr << "WARN: unknown GPU " << prop.name << ", using " << kernelMillis << "ms for kernel\n";
    }
  }

  /*
  Table 5
  512^3
  512^3 on Pascal 34.1ms
  512^3 on Volta  20.1ms
  */

  Method methods = Method::None;
  if (result["remote"].as<bool>()) {
    methods |= Method::CudaMpi;
  }
  if (result["colocated"].as<bool>()) {
    methods |= Method::ColoPackMemcpyUnpack;
  }
  if (result["peer"].as<bool>()) {
    methods |= Method::CudaMemcpyPeer;
  }
  if (result["kernel"].as<bool>()) {
    methods |= Method::CudaKernel;
  }
  if (Method::None == methods) {
    methods = Method::Default;
  }

  PlacementStrategy strategy = PlacementStrategy::NodeAware;
  if (result["trivial"].as<bool>()) {
    strategy = PlacementStrategy::Trivial;
  }

  if (0 == rank) {
    std::cout << "domain: " << x << "," << y << "," << z << "\n";
  }

  {
    size_t radius = 3;

    DistributedDomain dd(x, y, z);

    dd.set_methods(methods);
    dd.set_radius(radius);
    dd.set_placement(strategy);

    auto dh0 = dd.add_data<float>("d0");
    // auto dh1 = dd.add_data<float>("d1");
    // auto dh2 = dd.add_data<float>("d2");
    // auto dh3 = dd.add_data<float>("d3");

    dd.realize();

    MPI_Barrier(MPI_COMM_WORLD);

    // create a compute stream for each local domain
    std::vector<RcStream> computeStreams(dd.domains().size());
    for (size_t di = 0; di < dd.domains().size(); ++di) {
      computeStreams[di] = RcStream(dd.domains()[di].gpu());
    }

    std::cerr << "init\n";
    for (size_t di = 0; di < dd.domains().size(); ++di) {
      auto &d = dd.domains()[di];
      d.set_device();
      dim3 dimBlock = Dim3::make_block_dim(d.raw_size(), 512);
      dim3 dimGrid = ((d.raw_size()) + Dim3(dimBlock) - 1) / (Dim3(dimBlock));
      init_kernel<<<dimGrid, dimBlock, 0, computeStreams[di]>>>(d.get_curr_accessor(dh0), d.get_compute_region(), 10);
      CUDA_RUNTIME(hipDeviceSynchronize());
    }

    if (0)
      dd.write_paraview("init");

    const std::vector<Rect3> interiors = dd.get_interior();
    const std::vector<std::vector<Rect3>> exteriors = dd.get_exterior();

    for (size_t iter = 0; iter < 5; ++iter) {

      // launch operations on interior
      for (size_t di = 0; di < dd.domains().size(); ++di) {
        auto &d = dd.domains()[di];
        const Accessor<float> src0 = d.get_curr_accessor<float>(dh0);
        const Accessor<float> dst0 = d.get_next_accessor<float>(dh0);
        nvtxRangePush("launch");
        const Rect3 cr = interiors[di];
        std::cerr << rank << ": launch on region=" << cr << " (interior)\n";
        // std::cerr << src0.origin() << "=src0 origin\n";
        d.set_device();
        dim3 dimBlock = Dim3::make_block_dim(cr.hi - cr.lo, 512);
        dim3 dimGrid = ((cr.hi - cr.lo) + Dim3(dimBlock) - 1) / (Dim3(dimBlock));
        stencil_kernel<<<dimGrid, dimBlock, 0, computeStreams[di]>>>(dst0, src0, cr);
        CUDA_RUNTIME(hipGetLastError());
        nvtxRangePop(); // launch
                        // CUDA_RUNTIME(hipDeviceSynchronize());
      }

      // exchange halo
      std::cerr << rank << ": exchange\n";
      dd.exchange();

      // operate on exterior
      for (size_t di = 0; di < dd.domains().size(); ++di) {
        auto &d = dd.domains()[di];
        const Accessor<float> src0 = d.get_curr_accessor<float>(dh0);
        const Accessor<float> dst0 = d.get_next_accessor<float>(dh0);
        for (size_t si = 0; si < exteriors[di].size(); ++si) {
          nvtxRangePush("launch");
          const Rect3 cr = exteriors[di][si];
          std::cerr << rank << ": launch on region=" << cr << " (exterior)\n";
          // std::cerr << src0.origin() << "=src0 origin\n";
          d.set_device();
          dim3 dimBlock = Dim3::make_block_dim(cr.hi - cr.lo, 512);
          dim3 dimGrid = ((cr.hi - cr.lo) + Dim3(dimBlock) - 1) / (Dim3(dimBlock));
          stencil_kernel<<<dimGrid, dimBlock, 0, computeStreams[di]>>>(dst0, src0, cr);
          CUDA_RUNTIME(hipGetLastError());
          nvtxRangePop(); // launch
          // CUDA_RUNTIME(hipDeviceSynchronize());
        }
      }

      // wait for stencil to complete
      for (auto &s : computeStreams) {
        CUDA_RUNTIME(hipStreamSynchronize(s));
      }

      // swap
      dd.swap();
    }

    if (0)
      dd.write_paraview("final");

  } // send domains out of scope before MPI_Finalize

  MPI_Finalize();

  return 0;
}
