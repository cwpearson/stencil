#include "hip/hip_runtime.h"
#include "stencil/unpack_kernel.cuh"

__device__ void grid_unpack(void *__restrict__ dst, const Dim3 dstSize, const Dim3 dstPos, const Dim3 dstExtent,
                            const void *__restrict__ src, const size_t elemSize) {

  const size_t tz = blockDim.z * blockIdx.z + threadIdx.z;
  const size_t ty = blockDim.y * blockIdx.y + threadIdx.y;
  const size_t tx = blockDim.x * blockIdx.x + threadIdx.x;

  for (size_t zi = tz; zi < dstExtent.z; zi += blockDim.z * gridDim.z) {
    for (size_t yi = ty; yi < dstExtent.y; yi += blockDim.y * gridDim.y) {
      for (size_t xi = tx; xi < dstExtent.x; xi += blockDim.x * gridDim.x) {
        size_t zo = zi + dstPos.z;
        size_t yo = yi + dstPos.y;
        size_t xo = xi + dstPos.x;
        size_t oi = zo * dstSize.y * dstSize.x + yo * dstSize.x + xo;
        size_t ii = zi * dstExtent.y * dstExtent.x + yi * dstExtent.x + xi;
        // printf("%lu %lu %lu [%lu] -> %lu %lu %lu [%lu]\n", xi, yi, zi, ii,
        // xo,
        //        yo, zo, oi);
        if (4 == elemSize) {
          uint32_t *pDst = reinterpret_cast<uint32_t *>(dst);
          const uint32_t *pSrc = reinterpret_cast<const uint32_t *>(src);
          pDst[oi] = pSrc[ii];
        } else if (8 == elemSize) {
          uint64_t *pDst = reinterpret_cast<uint64_t *>(dst);
          const uint64_t *pSrc = reinterpret_cast<const uint64_t *>(src);
          pDst[oi] = pSrc[ii];
        } else {
          char *pDst = reinterpret_cast<char *>(dst);
          const char *pSrc = reinterpret_cast<const char *>(src);
          memcpy(&pDst[oi * elemSize], &pSrc[ii * elemSize], elemSize);
        }
      }
    }
  }
}

__global__ void unpack(void *__restrict__ dst, const Dim3 dstSize, const size_t dstPitch, const Dim3 dstPos,
                       const Dim3 dstExtent, const void *__restrict__ src, const size_t elemSize) {
  grid_unpack(dst, dstSize, dstPos, dstExtent, src, elemSize);
}