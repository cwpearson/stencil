#include "hip/hip_runtime.h"
#include <cassert>
#include <chrono>
#include <cmath>
#include <numeric>

#include "stencil/argparse.hpp"
#include "stencil/cuda_runtime.hpp"
#include "stencil/mat2d.hpp"

double (*ExchangeFunc)(const Mat2D<int64_t> &comm, const int nIters);

void exchange_cuda_memcpy_peer(double *rtotal, Mat2D<double> *rtimes, const Mat2D<int64_t> &comm, const int nIters) {

  // enable peer access
  for (size_t src = 0; src < comm.shape().y; ++src) {
    for (size_t dst = 0; dst < comm.shape().x; ++dst) {
      if (src == dst) {
        continue;
      } else {
        int canAccess;
        CUDA_RUNTIME(hipDeviceCanAccessPeer(&canAccess, src, dst));
        if (canAccess) {
          CUDA_RUNTIME(hipSetDevice(src));
          hipError_t err = hipDeviceEnablePeerAccess(dst, 0 /*flags*/);
          if (hipSuccess == err || hipErrorPeerAccessAlreadyEnabled == err) {
            hipGetLastError(); // clear the error
          } else if (hipErrorInvalidDevice == err) {
            hipGetLastError(); // clear the error
          } else {
            CUDA_RUNTIME(err);
          }
        } else {
        }
      }
      CUDA_RUNTIME(hipGetLastError());
    }
  }

  size_t nGpus = std::max(comm.shape().x, comm.shape().y);
  Mat2D<hipStream_t> streams(nGpus, nGpus, nullptr);
  Mat2D<hipEvent_t> startEvents(nGpus, nGpus, nullptr);
  Mat2D<hipEvent_t> stopEvents(nGpus, nGpus, nullptr);
  Mat2D<void *> srcBufs(nGpus, nGpus, nullptr);
  Mat2D<void *> dstBufs(nGpus, nGpus, nullptr);
  Mat2D<double> times(nGpus, nGpus, 0);
  for (size_t i = 0; i < nGpus; ++i) {
    for (size_t j = 0; j < nGpus; ++j) {
      CUDA_RUNTIME(hipSetDevice(i));
      CUDA_RUNTIME(hipStreamCreate(&streams.at(i, j)));
      CUDA_RUNTIME(hipEventCreate(&startEvents.at(i, j)));
      CUDA_RUNTIME(hipEventCreate(&stopEvents.at(i, j)));
      CUDA_RUNTIME(hipMalloc(&srcBufs.at(i, j), comm.at(i, j)));

      CUDA_RUNTIME(hipSetDevice(j));
      CUDA_RUNTIME(hipMalloc(&dstBufs.at(i, j), comm.at(i, j)));
    }
  }

  std::chrono::duration<double> elapsed = std::chrono::seconds(0);

  for (int n = 0; n < nIters; ++n) {

    auto start = std::chrono::system_clock::now();
    for (size_t i = 0; i < nGpus; ++i) {
      for (size_t j = 0; j < nGpus; ++j) {
        CUDA_RUNTIME(hipSetDevice(i));
        CUDA_RUNTIME(hipEventRecord(startEvents.at(i, j), streams.at(i, j)));
        CUDA_RUNTIME(hipMemcpyPeerAsync(dstBufs.at(i, j), j, srcBufs.at(i, j),
                                         i, comm.at(i, j), streams.at(i, j)));
        CUDA_RUNTIME(hipEventRecord(stopEvents.at(i, j), streams.at(i, j)));
  	}

    }

    for (size_t i = 0; i < nGpus; ++i) {
      for (size_t j = 0; j < nGpus; ++j) {
        CUDA_RUNTIME(hipStreamSynchronize(streams.at(i, j)));
      }
    }
    elapsed += std::chrono::system_clock::now() - start;

    // get time for each transfer
    for (size_t i = 0; i < nGpus; ++i) {
      for (size_t j = 0; j < nGpus; ++j) {
        float ms;
        CUDA_RUNTIME(hipEventElapsedTime(&ms, startEvents.at(i, j),
                                          stopEvents.at(i, j)));
        times.at(i, j) += ms / 1000.0;
      }
    }
  }

  // normalize times by nIters
  for (size_t i = 0; i < nGpus; ++i) {
    for (size_t j = 0; j < nGpus; ++j) {
      times.at(i, j) /= nIters;
    }
  }

  /*
  std::cout << "bw\n";
  for (size_t i = 0; i < nGpus; ++i) {
    for (size_t j = 0; j < nGpus; ++j) {
      printf("%.4e ", comm.at(i, j) / times.at(i, j));
    }
    std::cout << "\n";
  }
  std::cout << "time\n";
  for (size_t i = 0; i < nGpus; ++i) {
    for (size_t j = 0; j < nGpus; ++j) {
      printf("%.4e ", times.at(i, j));
    }
    std::cout << "\n";
  }
*/
  // free stuff
  for (size_t i = 0; i < nGpus; ++i) {
    for (size_t j = 0; j < nGpus; ++j) {
      CUDA_RUNTIME(hipStreamDestroy(streams.at(i, j)));
      CUDA_RUNTIME(hipEventDestroy(startEvents.at(i, j)));
      CUDA_RUNTIME(hipEventDestroy(stopEvents.at(i, j)));
      CUDA_RUNTIME(hipFree(srcBufs.at(i, j)));
      CUDA_RUNTIME(hipFree(dstBufs.at(i, j)));
    }
  }

  if (rtotal) *rtotal = elapsed.count() / double(nIters);
  if (rtimes) *rtimes = times;
}

int main(int argc, char **argv) {

  const int64_t K = 1024;
  const int64_t M = K * K;
  const int64_t G = K * K * K;

  Mat2D<int64_t> allToAll1G {
    {G, G, G, G},
    {G, G, G, G},
    {G, G, G, G},
    {G, G, G, G},
  };
  Mat2D<int64_t> allToAll8M {
    {8*M, 8*M, 8*M, 8*M},
    {8*M, 8*M, 8*M, 8*M},
    {8*M, 8*M, 8*M, 8*M},
    {8*M, 8*M, 8*M, 8*M},
  };
  Mat2D<int64_t> stencil512x256x512 {
    {12*M, 12*M, 6*M, 30*K},
    {12*M, 12*M, 30*K, 8*M},
    {6*M, 30*K, 12*M, 12*M},
    {30*K, 6*M, 12*M, 12*M},
  };
  Mat2D<int64_t> local1G {
    {G, G, 0, 0},
    {G, G, 0, 0},
    {0, 0, G, G},
    {0, 0, G, G},
  };
  Mat2D<int64_t> local1Gremote100M {
    {G, G, 100*M, 100*M},
    {G, G, 100*M, 100*M},
    {100*M, 100*M, G, G},
    {100*M, 100*M, G, G},
  };
  // clang-format on

  double time;

  std::cout << "stencil\n";
  exchange_cuda_memcpy_peer(&time, nullptr, stencil512x256x512, 30);
  std::cout << time << "\n";
  std::cout << "All-to-all 8MiB\n";
  exchange_cuda_memcpy_peer(&time, nullptr, allToAll8M, 30);
  std::cout << time << "\n";
  std::cout << "All-to-all 1GiB\n";
  exchange_cuda_memcpy_peer(&time, nullptr, allToAll1G, 30);
  std::cout << time << "\n";
  std::cout << "Local 1GiB\n";
  exchange_cuda_memcpy_peer(&time, nullptr, local1G, 30);
  std::cout << time << "\n";
  std::cout << "Local 1GiB Remote 100M\n";
  exchange_cuda_memcpy_peer(&time, nullptr, local1Gremote100M, 30);
  std::cout << time << "\n";
}
