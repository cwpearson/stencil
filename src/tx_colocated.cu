#include "stencil/tx_colocated.cuh"

#include "stencil/copy.cuh"

#include <nvToolsExt.h>
#include <nvToolsExtCudaRt.h>

#include <algorithm>

ColoHaloSender::ColoHaloSender(int srcRank, int srcDom, int dstRank, int dstDom, LocalDomain &domain,
                               Placement *placement)
    : srcRank_(srcRank), dstRank_(dstRank), srcDom_(srcDom), dstDom_(dstDom), domain_(&domain), placement_(placement),
      stream_(domain.gpu(), RcStream::Priority::HIGH), ipcSender_(srcRank, srcDom, dstRank, dstDom, domain.gpu()),
      translate_(nullptr), // derived class picks translator implementation
      dstDomCurrDatasDev_(nullptr) {
  std::string streamName("ColoHaloSender_");
  streamName += "r" + std::to_string(srcRank);
  streamName += "d" + std::to_string(srcDom);
  streamName += "->r" + std::to_string(dstRank);
  streamName += "d" + std::to_string(dstDom);
  nvtxNameCudaStreamA(stream_, streamName.c_str());
}

ColoHaloSender::~ColoHaloSender() {
  assert(domain_);
  CUDA_RUNTIME(hipSetDevice(domain_->gpu()));
  // free cuda malloc
  CUDA_RUNTIME(hipFree(dstDomCurrDatasDev_));
  dstDomCurrDatasDev_ = nullptr;

  // free mem handle
  for (hipPitchedPtr &p : dstDomCurrDatas_) {
    CUDA_RUNTIME(hipIpcCloseMemHandle(p.ptr));
  }
  dstDomCurrDatas_.clear();

  delete translate_;
  translate_ = nullptr;
}

void ColoHaloSender::start_prepare(const std::vector<Message> &outbox) {
  nvtxRangePush("ColoHaloSender::start_prepare");
  outbox_ = outbox;
  std::sort(outbox_.begin(), outbox_.end());
  // outbox should only have messages for our domain and the dst domain
  for (const Message &msg : outbox_) {
    assert(msg.srcGPU_ == srcDom_ && "outbox has a wrong message");
    assert(msg.dstGPU_ == dstDom_ && "outbox has a wrong message");
  }

  // Post recieve the memhandles for the destination buffers
  const int memHandleTag = make_tag<MsgKind::ColocatedMem>(ipc_tag_payload(srcDom_, dstDom_));
  memHandles_.resize(domain_->num_data());
  MPI_Irecv(memHandles_.data(), memHandles_.size() * sizeof(memHandles_[0]), MPI_BYTE, dstRank_, memHandleTag,
            MPI_COMM_WORLD, &memReq_);

  // Post recieve for the pitch information
  const int ptrHandleTag = make_tag<MsgKind::ColocatedPtr>(ipc_tag_payload(srcDom_, dstDom_));
  dstDomCurrDatas_.resize(domain_->num_data());
  MPI_Irecv(dstDomCurrDatas_.data(), dstDomCurrDatas_.size() * sizeof(dstDomCurrDatas_[0]), MPI_BYTE, dstRank_,
            ptrHandleTag, MPI_COMM_WORLD, &ptrReq_);

  ipcSender_.async_prepare();
  nvtxRangePop();
}

void ColoHaloSender::finish_prepare() {
  LOG_SPEW("ColoHaloSender::finish_prepare: waiting for mem handles...");
  // recieve mem handles
  MPI_Wait(&memReq_, MPI_STATUS_IGNORE);
  LOG_SPEW("ColoHaloSender::finish_prepare: got mem handles");

  // recieve pitch information
  LOG_SPEW("ColoHaloSender::finish_prepare: waiting for pitch information");
  MPI_Wait(&ptrReq_, MPI_STATUS_IGNORE);
  LOG_SPEW("ColoHaloSender::finish_prepare: got pitch info");

  // convert to pointers
  CUDA_RUNTIME(hipSetDevice(domain_->gpu()));
  for (size_t i = 0; i < memHandles_.size(); ++i) {
    void *ptr = nullptr;
    CUDA_RUNTIME(hipIpcOpenMemHandle(&ptr, memHandles_[i], hipIpcMemLazyEnablePeerAccess));
    dstDomCurrDatas_[i].ptr = ptr; // overwrite with ptr that is valid in this address space
  }
  LOG_SPEW("ColoHaloSender::finish_prepare: converted to pointers");

  // push pointers to device so they can be used in the kernel
  CUDA_RUNTIME(hipMalloc(&dstDomCurrDatasDev_, dstDomCurrDatas_.size() * sizeof(dstDomCurrDatas_[0])));
  CUDA_RUNTIME(hipMemcpy(dstDomCurrDatasDev_, dstDomCurrDatas_.data(),
                          dstDomCurrDatas_.size() * sizeof(dstDomCurrDatas_[0]), hipMemcpyHostToDevice));
  LOG_SPEW("ColoHaloSender::finish_prepare: pushed pointers");

  {
    std::vector<Translator::RegionParams> params;
    // get the dst idx;
    const Dim3 dstIdx = placement_->get_idx(dstRank_, dstDom_);

    for (const Message &msg : outbox_) {

      // the direction is not necessarily dst - src, since these domains could be neighbors in multiple directions
      // so use msg.dir_

      // determine the size of the destination
      const Dim3 dstSz = placement_->subdomain_size(dstIdx);
      const Dim3 dstPos = LocalDomain::halo_pos(msg.dir_ * -1, dstSz, domain_->radius(), true /*exterior*/);

      const Dim3 srcPos = domain_->halo_pos(msg.dir_, false /*interior*/);
      const Dim3 extent = domain_->halo_extent(msg.dir_);

      Translator::RegionParams p{.dstPtrs = dstDomCurrDatas_.data(),
                                 .dstPos = dstPos,
                                 .srcPtrs = domain_->curr_datas().data(),
                                 .srcPos = srcPos,
                                 .extent = extent,
                                 .elemSizes = domain_->elem_sizes().data(),
                                 .n = domain_->num_data()};
      params.push_back(p);
    }
    LOG_SPEW("ColoHaloSender::finish_prepare: cvt outbox to params");
    assert(translate_);
    translate_->prepare(params);
  }
  LOG_SPEW("ColoHaloSender::finish_prepare: prepared translator");

  ipcSender_.wait_prepare();
}

void ColoHaloSender::send() {
  nvtxRangePush("ColoHaloSender::send");
  assert(translate_);
  translate_->async(stream_);
  CUDA_RUNTIME(hipEventRecord(ipcSender_.event(), stream_));
  ipcSender_.async_notify();
  nvtxRangePop(); // ColoHaloSender::send
}

void ColoHaloSender::wait() {
  ipcSender_.wait_notify();
  CUDA_RUNTIME(hipEventSynchronize(ipcSender_.event()));
}

ColoMemcpy3dHaloSender::ColoMemcpy3dHaloSender(int srcRank, int srcDom, int dstRank, int dstDom, LocalDomain &domain,
                                               Placement *placement)
    : ColoHaloSender(srcRank, srcDom, dstRank, dstDom, domain, placement) {
  assert(!translate_);
  translate_ = new TranslatorMemcpy3D();
}

ColoQuantityKernelSender::ColoQuantityKernelSender(int srcRank, int srcDom, int dstRank, int dstDom, LocalDomain &domain,
                                   Placement *placement)
    : ColoHaloSender(srcRank, srcDom, dstRank, dstDom, domain, placement) {
  assert(!translate_);
  translate_ = new TranslatorKernel(domain.gpu());
}

ColoRegionKernelSender::ColoRegionKernelSender(int srcRank, int srcDom, int dstRank, int dstDom, LocalDomain &domain,
                                             Placement *placement)
    : ColoHaloSender(srcRank, srcDom, dstRank, dstDom, domain, placement) {
  assert(!translate_);
  translate_ = new TranslatorMultiKernel(domain.gpu());
}

ColoHaloRecver::ColoHaloRecver(int srcRank, int srcDom, int dstRank, int dstDom, LocalDomain &domain)
    : srcRank_(srcRank), srcDom_(srcDom), dstDom_(dstDom), domain_(&domain),
      stream_(domain.gpu(), RcStream::Priority::HIGH), ipcRecver_(srcRank, srcDom, dstRank, dstDom, domain.gpu()),
      state_(State::NONE) {}

ColoHaloRecver::~ColoHaloRecver() {}

void ColoHaloRecver::start_prepare(const std::vector<Message> &inbox) {

  // we don't do anything to recv messages
  (void)inbox;

  // convert quantity pointers to handles
  for (const hipPitchedPtr &p : domain_->curr_datas()) {
    hipIpcMemHandle_t handle;
    CUDA_RUNTIME(hipIpcGetMemHandle(&handle, p.ptr));
    handles_.push_back(handle);
  }

  // post send handles to source rank
  const int memTag = make_tag<MsgKind::ColocatedMem>(ipc_tag_payload(srcDom_, dstDom_));
  MPI_Isend(handles_.data(), handles_.size() * sizeof(handles_[0]), MPI_BYTE, srcRank_, memTag, MPI_COMM_WORLD,
            &memReq_);

  // post send of pitch information
  const int ptrTag = make_tag<MsgKind::ColocatedPtr>(ipc_tag_payload(srcDom_, dstDom_));
  MPI_Isend(domain_->curr_datas().data(), domain_->curr_datas().size() * sizeof(domain_->curr_datas()[0]), MPI_BYTE,
            srcRank_, ptrTag, MPI_COMM_WORLD, &ptrReq_);

  ipcRecver_.async_prepare();
}

void ColoHaloRecver::finish_prepare() {

  // wait for mem handles to be sent
  MPI_Wait(&memReq_, MPI_STATUS_IGNORE);

  ipcRecver_.wait_prepare();
}

void ColoHaloRecver::recv() {
  assert(State::NONE == state_);
  ipcRecver_.async_listen();
  state_ = State::WAIT_NOTIFY;
}

// once we are in the WAIT_KERNEL state, there's nothing else we need to do
bool ColoHaloRecver::active() { return state_ == State::WAIT_NOTIFY; }

bool ColoHaloRecver::next_ready() {
  if (State::WAIT_NOTIFY == state_) {
    return ipcRecver_.test_listen();
  } else { // should only be asked this in active() states
    LOG_FATAL("unexpected state");
  }
}

void ColoHaloRecver::next() {
  if (State::WAIT_NOTIFY == state_) {
    state_ = State::WAIT_KERNEL;
  }
}

void ColoHaloRecver::wait() {
  // wait on the event that the sender recorded after the kernel
  assert(stream_.device() == domain_->gpu());
  CUDA_RUNTIME(hipEventSynchronize(ipcRecver_.event()));
  state_ = State::NONE;
}
