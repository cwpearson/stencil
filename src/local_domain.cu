#include "stencil/local_domain.cuh"

#include <nvToolsExt.h>

LocalDomain::LocalDomain(Dim3 sz, Dim3 origin, int dev)
    : sz_(sz), origin_(origin), dev_(dev), devCurrDataPtrs_(nullptr), devDataElemSize_(nullptr) {}

LocalDomain::~LocalDomain() {
  CUDA_RUNTIME(hipGetLastError());

  CUDA_RUNTIME(hipSetDevice(dev_));
  for (auto p : currDataPtrs_) {
    if (p)
      CUDA_RUNTIME(hipFree(p));
  }
  if (devCurrDataPtrs_)
    CUDA_RUNTIME(hipFree(devCurrDataPtrs_));

  for (auto p : nextDataPtrs_) {
    if (p)
      CUDA_RUNTIME(hipFree(p));
  }
  if (devDataElemSize_)
    CUDA_RUNTIME(hipFree(devDataElemSize_));
  CUDA_RUNTIME(hipGetLastError());
}

void LocalDomain::set_device(CudaErrorsFatal fatal) {
  hipError_t err = hipSetDevice(dev_);
  if (CudaErrorsFatal::YES == fatal) {
    CUDA_RUNTIME(err)
  } else {
    (void)err;
  }
}

Rect3 LocalDomain::halo_coords(const Dim3 &dir, const bool halo) const {
  /* convert halo position as offset from allocation to coordinates
   */

  // get the offset of the halo from the allocation
  Dim3 pos = halo_pos(dir, halo);
  Dim3 ext = halo_extent(dir);

  // convert to offset from the origin
  // the size of the negative halo is equal to the size of the negative kernel radius
  // translate
  pos.z -= radius_.z(-1);
  pos.y -= radius_.y(-1);
  pos.x -= radius_.x(-1);

  // shift by the origin
  pos += origin_;

  return Rect3(pos, pos + ext);
}

Rect3 LocalDomain::get_compute_region() const noexcept {
  Dim3 lo = origin();
  Dim3 hi = origin() + size();
  return Rect3(lo, hi);
}

void LocalDomain::swap() noexcept {
  nvtxRangePush("swap");

  // swap the host copy of the pointers
  assert(currDataPtrs_.size() == nextDataPtrs_.size());
  for (size_t i = 0; i < currDataPtrs_.size(); ++i) {
    std::swap(currDataPtrs_[i], nextDataPtrs_[i]);
  }

  // update the device version of the pointers
  CUDA_RUNTIME(hipMemcpy(devCurrDataPtrs_, currDataPtrs_.data(), currDataPtrs_.size() * sizeof(currDataPtrs_[0]),
                          hipMemcpyHostToDevice));
  nvtxRangePop();
}

Dim3 LocalDomain::halo_pos(const Dim3 &dir, const bool halo) const noexcept {
  assert(dir.all_gt(-2));
  assert(dir.all_lt(2));

  Dim3 ret;

  // +xhalo is the left edge + -x radius + the interior
  // +x interior is just the left edge + interior size
  if (1 == dir.x) {
    ret.x = sz_.x + (halo ? radius_.x(-1) : 0);
  } else if (-1 == dir.x) {
    ret.x = halo ? 0 : radius_.x(-1);
  } else if (0 == dir.x) {
    ret.x = radius_.x(-1);
  } else {
    LOG_FATAL("unreachable");
  }

  if (1 == dir.y) {
    ret.y = sz_.y + (halo ? radius_.y(-1) : 0);
  } else if (-1 == dir.y) {
    ret.y = halo ? 0 : radius_.y(-1);
  } else if (0 == dir.y) {
    ret.y = radius_.y(-1);
  } else {
    LOG_FATAL("unreachable");
  }

  if (1 == dir.z) {
    ret.z = sz_.z + (halo ? radius_.z(-1) : 0);
  } else if (-1 == dir.z) {
    ret.z = halo ? 0 : radius_.z(-1);
  } else if (0 == dir.z) {
    ret.z = radius_.z(-1);
  } else {
    LOG_FATAL("unreachable");
  }

  return ret;
}

std::vector<unsigned char> LocalDomain::region_to_host(const Dim3 &pos, const Dim3 &ext,
                                                       const size_t qi // quantity index
                                                       ) const {

  const size_t bytes = elem_size(qi) * ext.flatten();

  // pack quantity
  CUDA_RUNTIME(hipSetDevice(gpu()));
  void *devBuf = nullptr;
  CUDA_RUNTIME(hipMalloc(&devBuf, bytes));
  const dim3 dimBlock = Dim3::make_block_dim(ext, 512);
  const dim3 dimGrid = (ext + Dim3(dimBlock) - 1) / (Dim3(dimBlock));
  pack_kernel<<<dimGrid, dimBlock>>>(devBuf, curr_data(qi), raw_size(), pos, ext, elem_size(qi));
  CUDA_RUNTIME(hipDeviceSynchronize());

  // copy quantity to host
  std::vector<unsigned char> hostBuf(bytes);
  CUDA_RUNTIME(hipMemcpy(hostBuf.data(), devBuf, hostBuf.size(), hipMemcpyDefault));

  float *ptr = reinterpret_cast<float *>(hostBuf.data());

  // free device buffer
  CUDA_RUNTIME(hipFree(devBuf));

  return hostBuf;
}

void LocalDomain::realize() {
  LOG_SPEW("in realize()");
  CUDA_RUNTIME(hipGetLastError());
  assert(currDataPtrs_.size() == nextDataPtrs_.size());
  assert(dataElemSize_.size() == nextDataPtrs_.size());

  LOG_INFO("origin is " << origin_);

  // allocate each data region
  CUDA_RUNTIME(hipSetDevice(dev_));
  // int rank;
  // MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  // std::cerr << "r" << rank << " dev=" << dev_ << "\n";
  for (int64_t i = 0; i < num_data(); ++i) {
    assert(i < dataElemSize_.size());
    int64_t elemSz = dataElemSize_[i];
    LOG_SPEW("elemSz=" << elemSz);
    LOG_SPEW("radius +x=" << radius_.x(1));
    LOG_SPEW("radius -x=" << radius_.x(-1));
    LOG_SPEW("radius +y=" << radius_.y(1));
    LOG_SPEW("radius -y=" << radius_.y(-1));
    LOG_SPEW("radius +z=" << radius_.z(1));
    LOG_SPEW("radius -z=" << radius_.z(-1));

    int64_t elemBytes = ((sz_.x + radius_.x(-1) + radius_.x(1)) * (sz_.y + radius_.y(-1) + radius_.y(1)) *
                         (sz_.z + radius_.z(-1) + radius_.z(1))) *
                        elemSz;
    LOG_SPEW("allocate " << elemBytes << " bytes");
    char *c = nullptr;
    char *n = nullptr;
    CUDA_RUNTIME(hipMalloc(&c, elemBytes));
    CUDA_RUNTIME(hipMalloc(&n, elemBytes));
    assert(uintptr_t(c) % elemSz == 0 && "allocation should be aligned");
    assert(uintptr_t(n) % elemSz == 0 && "allocation should be aligned");
    currDataPtrs_[i] = c;
    nextDataPtrs_[i] = n;
  }

  CUDA_RUNTIME(hipMalloc(&devCurrDataPtrs_, currDataPtrs_.size() * sizeof(currDataPtrs_[0])));
  CUDA_RUNTIME(hipMalloc(&devDataElemSize_, dataElemSize_.size() * sizeof(dataElemSize_[0])));
  CUDA_RUNTIME(hipMemcpy(devCurrDataPtrs_, currDataPtrs_.data(), currDataPtrs_.size() * sizeof(currDataPtrs_[0]),
                          hipMemcpyHostToDevice));
  CUDA_RUNTIME(hipMemcpy(devDataElemSize_, dataElemSize_.data(), dataElemSize_.size() * sizeof(dataElemSize_[0]),
                          hipMemcpyHostToDevice));
  CUDA_RUNTIME(hipGetLastError());
}