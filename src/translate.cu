#include "stencil/translate.cuh"

#include "stencil/copy.cuh"
#include "stencil/cuda_runtime.hpp"
#include "stencil/logging.hpp"
#include "stencil/rcstream.hpp"
#include "stencil/rt.hpp"

#include <hip/hip_runtime.h>

Translator::Translator() {
#ifdef STENCIL_USE_CUDA_GRAPH
  graph_ = 0;
  instance_ = 0;
#endif
}

Translator::~Translator() {
#ifdef STENCIL_USE_CUDA_GRAPH
  if (instance_) {
    CUDA_RUNTIME(hipGraphExecDestroy(instance_));
  }
  if (graph_) {
    CUDA_RUNTIME(hipGraphDestroy(graph_));
  }
#endif
}

void Translator::prepare(const std::vector<Params> &params) {

  LOG_SPEW("params.size()=" << params.size());

  // convert all Params into individual 3D copies
  for (const Params &ps : params) {
    LOG_SPEW("ps.n=" << ps.n);
    assert(ps.dsts);
    assert(ps.srcs);
    assert(ps.elemSizes);
    for (int64_t i = 0; i < ps.n; ++i) {
      Param p(ps.dsts[i], ps.dstPos, ps.srcs[i], ps.srcPos, ps.extent, ps.elemSizes[i]);
      params_.push_back(p);
    }
  }

// FIXME: this stream is valid on device 0, but the kernels in TDA::kernel are launched on device device_
// this may cause invalid resource handle for devices != 0
#ifdef STENCIL_USE_CUDA_GRAPH
  // create a stream to record from
  RcStream stream;

  CUDA_RUNTIME(hipStreamBeginCapture(stream, hipStreamCaptureModeThreadLocal));
  launch_all(stream);
  CUDA_RUNTIME(hipStreamEndCapture(stream, &graph_));
  CUDA_RUNTIME(hipGraphInstantiate(&instance_, graph_, NULL, NULL, 0));
#endif
}

void Translator::async(hipStream_t stream) {
#ifdef STENCIL_USE_CUDA_GRAPH
  CUDA_RUNTIME(rt::time(hipGraphLaunch, instance_, stream));
#else
  launch_all(stream);
#endif
}

TranslatorDirectAccess::TranslatorDirectAccess(int device) : Translator(), device_(device) {}

void TranslatorDirectAccess::launch_all(hipStream_t stream) {
  for (const Param &p : params_) {
    kernel(p, device_, stream);
  }
#ifdef STENCIL_USE_CUDA_GRAPH
  CUDA_RUNTIME(hipGetLastError());
#endif
}

void TranslatorDirectAccess::kernel(const Param &p, const int device, hipStream_t stream) {
  const dim3 dimBlock = Dim3::make_block_dim(p.extent, 512 /*threads per block*/);
  const dim3 dimGrid = (p.extent + Dim3(dimBlock) - 1) / (Dim3(dimBlock));
  CUDA_RUNTIME(hipSetDevice(device));
  LOG_SPEW("translate dev=" << device << " grid=" << dimGrid << " block=" << dimBlock);
  rt::launch(translate, dimGrid, dimBlock, 0, stream, p.dstPtr, p.dstPos, p.srcPtr, p.srcPos, p.extent, p.elemSize);
#ifndef STENCIL_USE_CUDA_GRAPH
  // 900: operation not permitted while stream is capturing
  CUDA_RUNTIME(rt::time(hipGetLastError));
#endif
}

void TranslatorMemcpy3D::launch_all(hipStream_t stream) {
  for (const Param &p : params_) {
    memcpy_3d_async(p, stream);
  }
}

void TranslatorMemcpy3D::memcpy_3d_async(const Param &param, hipStream_t stream) {
  hipMemcpy3DParms p = {};

  const uint64_t es = param.elemSize;

  // "offset into the src/dst objs in units of unsigned char"
  p.srcPos = make_hipPos(param.srcPos.x * es, param.srcPos.y, param.srcPos.z);
  p.dstPos = make_hipPos(param.dstPos.x * es, param.dstPos.y, param.dstPos.z);

  // "dimension of the transferred area in elements of unsigned char"
  p.extent = make_hipExtent(param.extent.x * es, param.extent.y, param.extent.z);

  // we mark our srcPtr as `const void*` since we will not modify data through it, but the cuda pitchedPtr is just
  // `void*`
  p.srcPtr = param.srcPtr;
  p.dstPtr = param.dstPtr;

  p.kind = hipMemcpyDeviceToDevice;
  LOG_SPEW("srcPtr.pitch " << p.srcPtr.pitch);
  LOG_SPEW("srcPtr.ptr " << p.srcPtr.ptr);
  LOG_SPEW("srcPos  [" << p.srcPos.x << "," << p.srcPos.y << "," << p.srcPos.z << "]");
  LOG_SPEW("dstPtr.pitch " << p.dstPtr.pitch);
  LOG_SPEW("dstPtr.ptr " << p.dstPtr.ptr);
  LOG_SPEW("dstPos  [" << p.dstPos.x << "," << p.dstPos.y << "," << p.dstPos.z << "]");
  LOG_SPEW("extent [dhw] = [" << p.extent.depth << "," << p.extent.height << "," << p.extent.width << "]");
  CUDA_RUNTIME(rt::time(hipMemcpy3DAsync, &p, stream));
}
