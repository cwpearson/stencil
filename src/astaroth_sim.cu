#include "hip/hip_runtime.h"
#include <chrono>
#include <cmath>
#include <thread>

#include <nvToolsExt.h>

#include <cxxopts/cxxopts.hpp>

#include "stencil/stencil.hpp"

/*! set dst[x,y,z] = sin(x + origin.x)
and halo to -1
*/
template <typename T>
__global__ void
init_kernel(T *dst,            //<! [out] pointer to beginning of dst allocation
            const Dim3 origin, //<! [in]
            const Dim3 rawSz,   //<! [in] 3D size of the dst and src allocations
            const double period //<! sin wave period
) {

  constexpr size_t radius = 1;
  const Dim3 domSz = rawSz - Dim3(2 * radius, 2 * radius, 2 * radius);

  const size_t gdz = gridDim.z;
  const size_t biz = blockIdx.z;
  const size_t bdz = blockDim.z;
  const size_t tiz = threadIdx.z;

  const size_t gdy = gridDim.y;
  const size_t biy = blockIdx.y;
  const size_t bdy = blockDim.y;
  const size_t tiy = threadIdx.y;

  const size_t gdx = gridDim.x;
  const size_t bix = blockIdx.x;
  const size_t bdx = blockDim.x;
  const size_t tix = threadIdx.x;

#ifndef _at
#define _at(arr, _x, _y, _z) arr[_z * rawSz.y * rawSz.x + _y * rawSz.x + _x]
#else
#error "_at already defined"
#endif

  for (size_t z = biz * bdz + tiz; z < rawSz.z; z += gdz * bdz) {
    for (size_t y = biy * bdy + tiy; y < rawSz.y; y += gdy * bdy) {
      for (size_t x = bix * bdx + tix; x < rawSz.x; x += gdx * bdx) {

        if (z >= radius && x >= radius && y >= radius && z < rawSz.z - radius &&
            y < rawSz.y - radius && x < rawSz.x - radius) {
          _at(dst, x, y, z) =
              sin((origin.x + x - radius) * 2 * 3.14159/ period);
        } else {
          _at(dst, x, y, z) = -10;
        }
      }
    }
  }

#undef _at
}




int main(int argc, char **argv) {

  cxxopts::Options options("MyProgram", "One line description of MyProgram");
  // clang-format off
  options.add_options()
  ("h,help", "Show help")
  ("remote", "Enable RemoteSender/Recver")
  ("cuda-aware-mpi", "Enable CudaAwareMpiSender/Recver")
  ("colocated", "Enable ColocatedHaloSender/Recver")
  ("peer", "Enable PeerAccessSender")
  ("kernel", "Enable PeerCopySender")
  ("trivial", "Skip node-aware placement")
  ("x", "x dim", cxxopts::value<int>()->default_value("512"))
  ("y", "y dim", cxxopts::value<int>()->default_value("512"))
  ("z", "z dim", cxxopts::value<int>()->default_value("512"))
  ("f,file", "File name", cxxopts::value<std::string>());
  // clang-format on

  auto result = options.parse(argc, argv);

  if (result["help"].as<bool>()) {
    std::cerr << options.help();
    exit(EXIT_SUCCESS);
  }

  MPI_Init(&argc, &argv);

  int size;
  int rank;
  MPI_Comm_size(MPI_COMM_WORLD, &size);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);

  int devCount;
  CUDA_RUNTIME(hipGetDeviceCount(&devCount));

  int numSubdoms;
  {
    MpiTopology topo(MPI_COMM_WORLD);
    numSubdoms = size / topo.colocated_size() * devCount;
  }

  if (0 == rank) {
    std::cout << "assuming " << numSubdoms << " subdomains\n";
  }

  double kernelMillis = 50;
  size_t x = result["x"].as<int>();
  size_t y = result["y"].as<int>(); 
  size_t z = result["z"].as<int>();

  hipDeviceProp_t prop;
  CUDA_RUNTIME(hipGetDeviceProperties(&prop, 0));
  if (std::string("Tesla V100-SXM2-32GB") == prop.name) {
    kernelMillis = 20.1;
  } else if (std::string("Tesla P100-SXM2-16GB") == prop.name) {
    kernelMillis = 34.1;
  } else {
    if (0 == rank) {
      std::cerr << "WARN: unknown GPU " << prop.name << ", using "
                << kernelMillis << "ms for kernel\n";
    }
  }

  /*
  Table 5
  512^3
  512^3 on Pascal 34.1ms
  512^3 on Volta  20.1ms
  */

  MethodFlags methods = MethodFlags::None;
  if (result["remote"].as<bool>()) {
    methods |= MethodFlags::CudaMpi;
  }
  if (result["cuda-aware-mpi"].as<bool>()) {
    methods |= MethodFlags::CudaAwareMpi;
  }
  if (result["colocated"].as<bool>()) {
    methods |= MethodFlags::CudaMpiColocated;
  }
  if (result["peer"].as<bool>()) {
    methods |= MethodFlags::CudaMemcpyPeer;
  }
  if (result["kernel"].as<bool>()) {
    methods |= MethodFlags::CudaKernel;
  }
  if (MethodFlags::None == methods) {
    methods = MethodFlags::All;
  }

  PlacementStrategy strategy = PlacementStrategy::NodeAware;
  if (result["trivial"].as<bool>()) {
    strategy = PlacementStrategy::Trivial;
  }

  if (0 == rank) {
    std::cout << "domain: " << x << "," << y << "," << z << "\n";
  }

  {
    size_t radius = 3;

    DistributedDomain dd(x, y, z);

    dd.set_methods(methods);
    dd.set_radius(radius);
    dd.set_placement(strategy);

    auto dh0 = dd.add_data<float>("d0");
    // auto dh1 = dd.add_data<float>("d1");
    // auto dh2 = dd.add_data<float>("d2");
    // auto dh3 = dd.add_data<float>("d3");

    dd.realize();

    MPI_Barrier(MPI_COMM_WORLD);

    std::cerr << "init\n";
    dim3 dimGrid(10, 10, 10);
    dim3 dimBlock(8, 8, 8);
    for (size_t di = 0; di < dd.domains().size(); ++di) {
      auto &d = dd.domains()[di];
      CUDA_RUNTIME(hipSetDevice(d.gpu()));
      init_kernel<<<dimGrid, dimBlock>>>(d.get_curr(dh0), d.origin(),
                                         d.raw_size(), 10);
      CUDA_RUNTIME(hipDeviceSynchronize());
    }

    dd.write_paraview("init");

    for (size_t iter = 0; iter < 5; ++iter) {
      std::cerr << "exchange\n";
      nvtxRangePush("exchange");
      dd.exchange();
      dd.swap();
      nvtxRangePop();

      std::cerr << "kernels\n";
      nvtxRangePush("kernels");
      auto dur = std::chrono::duration<double, std::milli>(kernelMillis);
      std::this_thread::sleep_for(dur);
      nvtxRangePop();
    }

    dd.write_paraview("final");

  } // send domains out of scope before MPI_Finalize

  MPI_Finalize();

  return 0;
}
