#include <chrono>
#include <cmath>
#include <thread>

#include <nvToolsExt.h>

#include <cxxopts/cxxopts.hpp>

#include "stencil/stencil.hpp"

int main(int argc, char **argv) {

  cxxopts::Options options("MyProgram", "One line description of MyProgram");
  // clang-format off
  options.add_options()
  ("h,help", "Show help")
  ("remote", "Enable RemoteSender/Recver")
  ("cuda-aware-mpi", "Enable CudaAwareMpiSender/Recver")
  ("colocated", "Enable ColocatedHaloSender/Recver")
  ("peer", "Enable PeerAccessSender")
  ("kernel", "Enable PeerCopySender")
  ("trivial", "Skip node-aware placement")
  ("x", "x dim", cxxopts::value<int>()->default_value("512"))
  ("y", "y dim", cxxopts::value<int>()->default_value("512"))
  ("z", "z dim", cxxopts::value<int>()->default_value("512"))
  ("f,file", "File name", cxxopts::value<std::string>());
  // clang-format on

  auto result = options.parse(argc, argv);

  if (result["help"].as<bool>()) {
    std::cerr << options.help();
    exit(EXIT_SUCCESS);
  }

  MPI_Init(&argc, &argv);

  int size;
  int rank;
  MPI_Comm_size(MPI_COMM_WORLD, &size);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);

  int devCount;
  CUDA_RUNTIME(hipGetDeviceCount(&devCount));

  int numSubdoms;
  {
    MpiTopology topo(MPI_COMM_WORLD);
    numSubdoms = size / topo.colocated_size() * devCount;
  }

  if (0 == rank) {
    std::cout << "assuming " << numSubdoms << " subdomains\n";
  }

  double kernelMillis = 50;
  size_t x = result["x"].as<int>();
  size_t y = result["y"].as<int>(); 
  size_t z = result["z"].as<int>();

  hipDeviceProp_t prop;
  CUDA_RUNTIME(hipGetDeviceProperties(&prop, 0));
  if (std::string("Tesla V100-SXM2-32GB") == prop.name) {
    kernelMillis = 20.1;
  } else if (std::string("Tesla P100-SXM2-16GB") == prop.name) {
    kernelMillis = 34.1;
  } else {
    if (0 == rank) {
      std::cerr << "WARN: unknown GPU " << prop.name << ", using "
                << kernelMillis << "ms for kernel\n";
    }
  }

  /*
  Table 5
  512^3
  512^3 on Pascal 34.1ms
  512^3 on Volta  20.1ms
  */

  MethodFlags methods = MethodFlags::None;
  if (result["remote"].as<bool>()) {
    methods |= MethodFlags::CudaMpi;
  }
  if (result["cuda-aware-mpi"].as<bool>()) {
    methods |= MethodFlags::CudaMpi;
  }
  if (result["colocated"].as<bool>()) {
    methods |= MethodFlags::CudaMpiColocated;
  }
  if (result["peer"].as<bool>()) {
    methods |= MethodFlags::CudaMemcpyPeer;
  }
  if (result["kernel"].as<bool>()) {
    methods |= MethodFlags::CudaKernel;
  }
  if (MethodFlags::None == methods) {
    methods = MethodFlags::All;
  }

  PlacementStrategy strategy = PlacementStrategy::NodeAware;
  if (result["trivial"].as<bool>()) {
    strategy = PlacementStrategy::Trivial;
  }

  if (0 == rank) {
    std::cout << "domain: " << x << "," << y << "," << z << "\n";
  }

  {
    size_t radius = 3;

    DistributedDomain dd(x, y, z);

    dd.set_methods(methods);
    dd.set_radius(radius);
    dd.set_placement(strategy);

    dd.add_data<float>();
    dd.add_data<float>();
    dd.add_data<float>();
    dd.add_data<float>();

    dd.realize();

    MPI_Barrier(MPI_COMM_WORLD);

    for (size_t iter = 0; iter < 5; ++iter) {
      std::cerr << "exchange\n";
      nvtxRangePush("exchange");
      dd.exchange();
      nvtxRangePop();

      std::cerr << "kernels\n";
      nvtxRangePush("kernels");
      auto dur = std::chrono::duration<double, std::milli>(kernelMillis);
      std::this_thread::sleep_for(dur);
      nvtxRangePop();
    }
  } // send domains out of scope before MPI_Finalize

  MPI_Finalize();

  return 0;
}
