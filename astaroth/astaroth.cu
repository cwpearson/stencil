#include "hip/hip_runtime.h"
/* 
Try to do some rough approximation of astaroth using the stencil library.
*/

#include <chrono>
#include <cmath>
#include <thread>

#include <nvToolsExt.h>

#include "argparse/argparse.hpp"
#include "stencil/stencil.hpp"


#include "kernels.h"

/*! set compute region to dst[x,y,z] = sin(x+y+z + origin.x + origin.y + origin.z)
 */
template <typename T>
__global__ void init_kernel(Accessor<T> dst,    //<! [out] pointer to beginning of allocation
                            const Rect3 cr,     //<! [in] compute region
                            const double period //<! [in] sine wave period
) {
  for (int64_t z = cr.lo.z + blockIdx.z * blockDim.z + threadIdx.z; z < cr.hi.z; z += gridDim.z * blockDim.z) {
    for (int64_t y = cr.lo.y + blockIdx.y * blockDim.y + threadIdx.y; y < cr.hi.y; y += gridDim.y * blockDim.y) {
      for (int64_t x = cr.lo.x + blockIdx.x * blockDim.x + threadIdx.x; x < cr.hi.x; x += gridDim.x * blockDim.x) {
        dst[Dim3(x, y, z)] = sin(2 * 3.14159 / period * x + 2 * 3.14159 / period * y + 2 * 3.14159 / period * z);
      }
    }
  }
}

/* Apply the stencil to the coordinates in `reg`
 */
__global__ void stencil_kernel(Accessor<AcReal> dst, const Accessor<AcReal> src, const Rect3 reg) {

  for (int64_t z = reg.lo.z + blockIdx.z * blockDim.z + threadIdx.z; z < reg.hi.z; z += gridDim.z * blockDim.z) {
    for (int64_t y = reg.lo.y + blockIdx.y * blockDim.y + threadIdx.y; y < reg.hi.y; y += gridDim.y * blockDim.y) {
      for (int64_t x = reg.lo.x + blockIdx.x * blockDim.x + threadIdx.x; x < reg.hi.x; x += gridDim.x * blockDim.x) {
        Dim3 o(x, y, z);
        float val = 0;
        val += src[o + Dim3(-1, 0, 0)];
        val += src[o + Dim3(0, -1, 0)];
        val += src[o + Dim3(0, 0, -1)];
        val += src[o + Dim3(1, 0, 0)];
        val += src[o + Dim3(0, 1, 0)];
        val += src[o + Dim3(0, 0, 1)];
        val /= 6;
        dst[o] = val;
      }
    }
  }
}

int main(int argc, char **argv) {

  MPI_Init(&argc, &argv);
  int size;
  int rank;
  MPI_Comm_size(MPI_COMM_WORLD, &size);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);

  argparse::Parser p("an astaroth-like performance simulator");

  int x = 256;
  int y = 256;
  int z = 256;
  bool trivialPlacement = false;

  p.add_flag(trivialPlacement, "--trivial")->help("use trivial placement");
  p.add_positional(x)->required();
  p.add_positional(y)->required();
  p.add_positional(z)->required();

  // If there was an error during parsing, report it.
  if (!p.parse(argc, argv)) {
    if (0 == rank) {
      std::cerr << p.help();
    }
    exit(EXIT_FAILURE);
  }

  if (p.need_help()) {
    if (0 == rank) {
      std::cerr << p.help();
    }
    exit(EXIT_SUCCESS);
  }

  int devCount;
  CUDA_RUNTIME(hipGetDeviceCount(&devCount));

  int numSubdoms;
  {
    MpiTopology topo(MPI_COMM_WORLD);
    numSubdoms = size / topo.colocated_size() * devCount;
  }

  if (0 == rank) {
    std::cout << "assuming " << numSubdoms << " subdomains\n";
  }

  Method methods = Method::None;
  methods |= Method::CudaMpi;
  methods |= Method::ColoPackMemcpyUnpack;
  methods |= Method::CudaMemcpyPeer;
  methods |= Method::CudaKernel;
  if (Method::None == methods) {
    methods = Method::Default;
  }

  PlacementStrategy strategy = PlacementStrategy::NodeAware;
  if (trivialPlacement) {
    strategy = PlacementStrategy::Trivial;
  }

  if (0 == rank) {
    std::cout << "domain: " << x << "," << y << "," << z << "\n";
  }

  {
    size_t radius = 3;

    DistributedDomain dd(x, y, z);

    dd.set_methods(methods);
    dd.set_radius(radius);
    dd.set_placement(strategy);

    auto dh0 = dd.add_data<AcReal>("d0");
    auto dh1 = dd.add_data<AcReal>("d1");
    auto dh2 = dd.add_data<AcReal>("d2");
    auto dh3 = dd.add_data<AcReal>("d3");
    auto dh4 = dd.add_data<AcReal>("d4");
    auto dh5 = dd.add_data<AcReal>("d5");
    auto dh6 = dd.add_data<AcReal>("d6");
    auto dh7 = dd.add_data<AcReal>("d7");

    dd.realize();

    MPI_Barrier(MPI_COMM_WORLD);

    // create a stream for the integration kernels to run in
    std::vector<RcStream> computeStreams(dd.domains().size());
    for (size_t di = 0; di < dd.domains().size(); ++di) {
      computeStreams[di] = RcStream(dd.domains()[di].gpu());
    }

    std::cerr << "init\n";
    for (size_t di = 0; di < dd.domains().size(); ++di) {
      auto &d = dd.domains()[di];
      d.set_device();
      dim3 dimBlock = Dim3::make_block_dim(d.raw_size(), 512);
      dim3 dimGrid = ((d.raw_size()) + Dim3(dimBlock) - 1) / (Dim3(dimBlock));
      init_kernel<<<dimGrid, dimBlock, 0, computeStreams[di]>>>(d.get_curr_accessor(dh0), d.get_compute_region(), 10);
      CUDA_RUNTIME(hipDeviceSynchronize());
    }

    if (0)
      dd.write_paraview("init");

    const std::vector<Rect3> interiors = dd.get_interior();
    const std::vector<std::vector<Rect3>> exteriors = dd.get_exterior();

    for (size_t iter = 0; iter < 5; ++iter) {

      // launch operations on interior
      for (size_t di = 0; di < dd.domains().size(); ++di) {
        auto &d = dd.domains()[di];
        const Accessor<AcReal> src0 = d.get_curr_accessor<AcReal>(dh0);
        const Accessor<AcReal> dst0 = d.get_next_accessor<AcReal>(dh0);
        nvtxRangePush("launch");
        const Rect3 cr = interiors[di];
        std::cerr << rank << ": launch on region=" << cr << " (interior)\n";
        // std::cerr << src0.origin() << "=src0 origin\n";
        d.set_device();
        dim3 dimBlock = Dim3::make_block_dim(cr.hi - cr.lo, 512);
        dim3 dimGrid = ((cr.hi - cr.lo) + Dim3(dimBlock) - 1) / (Dim3(dimBlock));
        stencil_kernel<<<dimGrid, dimBlock, 0, computeStreams[di]>>>(dst0, src0, cr);
        CUDA_RUNTIME(hipGetLastError());
        nvtxRangePop(); // launch
                        // CUDA_RUNTIME(hipDeviceSynchronize());
      }

      // exchange halo
      std::cerr << rank << ": exchange\n";
      dd.exchange();

      // operate on exterior
      for (size_t di = 0; di < dd.domains().size(); ++di) {
        auto &d = dd.domains()[di];
        const Accessor<AcReal> src0 = d.get_curr_accessor<AcReal>(dh0);
        const Accessor<AcReal> dst0 = d.get_next_accessor<AcReal>(dh0);
        for (size_t si = 0; si < exteriors[di].size(); ++si) {
          nvtxRangePush("launch");
          const Rect3 cr = exteriors[di][si];
          std::cerr << rank << ": launch on region=" << cr << " (exterior)\n";
          // std::cerr << src0.origin() << "=src0 origin\n";
          d.set_device();
          dim3 dimBlock = Dim3::make_block_dim(cr.hi - cr.lo, 512);
          dim3 dimGrid = ((cr.hi - cr.lo) + Dim3(dimBlock) - 1) / (Dim3(dimBlock));
          stencil_kernel<<<dimGrid, dimBlock, 0, computeStreams[di]>>>(dst0, src0, cr);
          CUDA_RUNTIME(hipGetLastError());
          nvtxRangePop(); // launch
          // CUDA_RUNTIME(hipDeviceSynchronize());
        }
      }

      // wait for stencil to complete
      for (auto &s : computeStreams) {
        CUDA_RUNTIME(hipStreamSynchronize(s));
      }

      // swap
      dd.swap();
    }

    if (0)
      dd.write_paraview("final");

  } // send domains out of scope before MPI_Finalize

  MPI_Finalize();

  return 0;
}
