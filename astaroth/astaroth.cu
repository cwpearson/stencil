#include "hip/hip_runtime.h"
/*
Try to do some rough approximation of astaroth using the stencil library.
*/

#include <chrono>
#include <cmath>
#include <thread>

#include <nvToolsExt.h>

#include "argparse/argparse.hpp"
#include "stencil/stencil.hpp"

#include "kernels.h"
#include "astaroth_utils.h"

#if 0
/*! set compute region to dst[x,y,z] = sin(x+y+z + origin.x + origin.y + origin.z)
 */
template <typename T>
__global__ void init_kernel(Accessor<T> dst,    //<! [out] pointer to beginning of allocation
                            const Rect3 cr,     //<! [in] compute region
                            const double period //<! [in] sine wave period
) {
  for (int64_t z = cr.lo.z + blockIdx.z * blockDim.z + threadIdx.z; z < cr.hi.z; z += gridDim.z * blockDim.z) {
    for (int64_t y = cr.lo.y + blockIdx.y * blockDim.y + threadIdx.y; y < cr.hi.y; y += gridDim.y * blockDim.y) {
      for (int64_t x = cr.lo.x + blockIdx.x * blockDim.x + threadIdx.x; x < cr.hi.x; x += gridDim.x * blockDim.x) {
        dst[Dim3(x, y, z)] = sin(2 * 3.14159 / period * x + 2 * 3.14159 / period * y + 2 * 3.14159 / period * z);
      }
    }
  }
}
#endif

#if 0
/* Apply the stencil to the coordinates in `reg`
 */
__global__ void stencil_kernel(Accessor<AcReal> dst, const Accessor<AcReal> src, const Rect3 reg) {

  for (int64_t z = reg.lo.z + blockIdx.z * blockDim.z + threadIdx.z; z < reg.hi.z; z += gridDim.z * blockDim.z) {
    for (int64_t y = reg.lo.y + blockIdx.y * blockDim.y + threadIdx.y; y < reg.hi.y; y += gridDim.y * blockDim.y) {
      for (int64_t x = reg.lo.x + blockIdx.x * blockDim.x + threadIdx.x; x < reg.hi.x; x += gridDim.x * blockDim.x) {
        Dim3 o(x, y, z);
        float val = 0;
        val += src[o + Dim3(-1, 0, 0)];
        val += src[o + Dim3(0, -1, 0)];
        val += src[o + Dim3(0, 0, -1)];
        val += src[o + Dim3(1, 0, 0)];
        val += src[o + Dim3(0, 1, 0)];
        val += src[o + Dim3(0, 0, 1)];
        val /= 6;
        dst[o] = val;
      }
    }
  }
}
#endif

int main(int argc, char **argv) {

  MPI_Init(&argc, &argv);
  int size;
  int rank;
  MPI_Comm_size(MPI_COMM_WORLD, &size);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);

  argparse::Parser p("an astaroth-like performance simulator");

  int x = 256;
  int y = 256;
  int z = 256;
  bool trivialPlacement = false;

  p.add_flag(trivialPlacement, "--trivial")->help("use trivial placement");
  p.add_positional(x)->required();
  p.add_positional(y)->required();
  p.add_positional(z)->required();

  // If there was an error during parsing, report it.
  if (!p.parse(argc, argv)) {
    if (0 == rank) {
      std::cerr << p.help();
    }
    exit(EXIT_FAILURE);
  }

  if (p.need_help()) {
    if (0 == rank) {
      std::cerr << p.help();
    }
    exit(EXIT_SUCCESS);
  }

  int devCount;
  CUDA_RUNTIME(hipGetDeviceCount(&devCount));

  int numSubdoms;
  {
    MpiTopology topo(MPI_COMM_WORLD);
    numSubdoms = size / topo.colocated_size() * devCount;
  }

  if (0 == rank) {
    std::cout << "assuming " << numSubdoms << " subdomains\n";
  }

  Method methods = Method::None;
  methods |= Method::CudaMpi;
  methods |= Method::ColoPackMemcpyUnpack;
  methods |= Method::CudaMemcpyPeer;
  methods |= Method::CudaKernel;
  if (Method::None == methods) {
    methods = Method::Default;
  }

  PlacementStrategy strategy = PlacementStrategy::NodeAware;
  if (trivialPlacement) {
    strategy = PlacementStrategy::Trivial;
  }

  if (0 == rank) {
    std::cout << "domain: " << x << "," << y << "," << z << "\n";
  }

  {
    size_t radius = 3;

    DistributedDomain dd(x, y, z);

    dd.set_methods(methods);
    dd.set_radius(radius);
    dd.set_placement(strategy);

    // add required data
    std::vector<DataHandle<AcReal>> handles;
    for (size_t i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
      handles.push_back(dd.add_data<AcReal>(""));
    }

    // create arrays
    std::cerr << "realize\n";
    dd.realize();

    MPI_Barrier(MPI_COMM_WORLD);

    // create a stream for the integration kernels to run in
    std::vector<RcStream> computeStreams(dd.domains().size());
    for (size_t di = 0; di < dd.domains().size(); ++di) {
      computeStreams[di] = RcStream(dd.domains()[di].gpu());
    }

    // create mesh info for each device
    for (size_t di = 0; di < dd.domains().size(); ++di) {
      int device = dd.domains()[di].gpu();
      acDeviceLoadDefaultUniforms(device);

     

      // like from config_loader.cc
      AcMeshInfo info{};
      acLoadConfig(AC_DEFAULT_CONFIG, &info);
      info.int_params[AC_nx];
      info.int_params[AC_nx];
      info.int_params[AC_nx];
      acDeviceLoadMeshInfo(device, info);
    }

    // create the VBAs for each domain
    std::vector<VertexBufferArray> vbas(dd.domains().size());
    for (size_t di = 0; di < dd.domains().size(); ++di) {
      VertexBufferArray &vba = vbas[di];
      LocalDomain &d = dd.domains()[di];

      for (size_t i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
        vba.in[i] = d.get_curr_accessor<AcReal>(handles[i]).ptr().ptr;
        vba.out[i] = d.get_next_accessor<AcReal>(handles[i]).ptr().ptr;
      }
    }

#if 0
    std::cerr << "init\n";
    for (size_t di = 0; di < dd.domains().size(); ++di) {
      auto &d = dd.domains()[di];
      d.set_device();
      dim3 dimBlock = Dim3::make_block_dim(d.raw_size(), 512);
      dim3 dimGrid = ((d.raw_size()) + Dim3(dimBlock) - 1) / (Dim3(dimBlock));
      init_kernel<<<dimGrid, dimBlock, 0, computeStreams[di]>>>(d.get_curr_accessor(dh0), d.get_compute_region(), 10);
      CUDA_RUNTIME(hipDeviceSynchronize());
    }

    if (0)
      dd.write_paraview("init");
#endif

    const std::vector<Rect3> interiors = dd.get_interior();
    const std::vector<std::vector<Rect3>> exteriors = dd.get_exterior();

    for (size_t iter = 0; iter < 5; ++iter) {

      // launch operations on interior
      for (size_t di = 0; di < dd.domains().size(); ++di) {
        auto &d = dd.domains()[di];
        nvtxRangePush("launch");
        const Rect3 cr = interiors[di];
        std::cerr << rank << ": launch on region=" << cr << " (interior)\n";
        // std::cerr << src0.origin() << "=src0 origin\n";
        d.set_device();
        integrate_substep(0, computeStreams[di], cr, vbas[di]);
        integrate_substep(1, computeStreams[di], cr, vbas[di]);
        integrate_substep(2, computeStreams[di], cr, vbas[di]);
        nvtxRangePop(); // launch
      }

      // exchange halo
      std::cerr << rank << ": exchange\n";
      dd.exchange();

      // launch on exteriors
      for (size_t di = 0; di < dd.domains().size(); ++di) {
        auto &d = dd.domains()[di];
        for (size_t si = 0; si < exteriors[di].size(); ++si) {
          nvtxRangePush("launch");
          const Rect3 cr = exteriors[di][si];
          std::cerr << rank << ": launch on region=" << cr << " (exterior)\n";
          // std::cerr << src0.origin() << "=src0 origin\n";
          d.set_device();
          integrate_substep(0, computeStreams[di], cr, vbas[di]);
          integrate_substep(1, computeStreams[di], cr, vbas[di]);
          integrate_substep(2, computeStreams[di], cr, vbas[di]);
          nvtxRangePop(); // launch
          // CUDA_RUNTIME(hipDeviceSynchronize());
        }
      }

      // wait for stencil to complete
      for (auto &s : computeStreams) {
        CUDA_RUNTIME(hipStreamSynchronize(s));
      }

      // swap
      dd.swap();
    }

    if (0)
      dd.write_paraview("final");

  } // send domains out of scope before MPI_Finalize

  MPI_Finalize();

  return 0;
}
