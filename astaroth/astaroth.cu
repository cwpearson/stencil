#include "hip/hip_runtime.h"
/*
Try to do some rough approximation of astaroth using the stencil library.
*/

#include <cmath>
#include <thread>

#include <nvToolsExt.h>

#include "argparse/argparse.hpp"
#include "stencil/stencil.hpp"

#include "astaroth_utils.h"
#include "kernels.h"
#include "statistics.hpp"

int3 decompose(int p) {

  int3 ret{1, 1, 1};

  for (int pf : prime_factors(p)) {
    if (ret.x <= ret.y && ret.x <= ret.z) {
      ret.x *= pf;
    } else if (ret.y <= ret.z) {
      ret.y *= pf;
    } else {
      ret.z *= pf;
    }
  }
  return ret;
}

int main(int argc, char **argv) {

  MPI_Init(&argc, &argv);
  int size;
  int rank;
  MPI_Comm_size(MPI_COMM_WORLD, &size);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);

  argparse::Parser p("Astaroth simulator");
  bool trivialPlacement = false;

  p.add_flag(trivialPlacement, "--trivial")->help("use trivial placement");

  // If there was an error during parsing, report it.
  if (!p.parse(argc, argv)) {
    if (0 == rank) {
      std::cerr << p.help();
    }
    MPI_Finalize();
    exit(EXIT_FAILURE);
  }

  if (p.need_help()) {
    if (0 == rank) {
      std::cerr << p.help();
    }
    MPI_Finalize();
    exit(EXIT_SUCCESS);
  }

  int devCount;
  CUDA_RUNTIME(hipGetDeviceCount(&devCount));

  int numSubdoms;
  {
    MpiTopology topo(MPI_COMM_WORLD);
    numSubdoms = size / topo.colocated_size() * devCount;
  }

  if (0 == rank) {
    std::cerr << "assuming " << numSubdoms << " subdomains\n";
  }

  // load config
  // like from config_loader.cc
  AcMeshInfo info{};
  acLoadConfig(AC_DEFAULT_CONFIG, &info);

  // figure out the whole domain size
  {
    int3 i3 = decompose(size);
    info.int_params[AC_nx] *= i3.x;
    info.int_params[AC_ny] *= i3.y;
    info.int_params[AC_nz] *= i3.z;
  }

  if (0 == rank) {
    std::cerr << "AC_nx=" << info.int_params[AC_nx] << "\n";
    std::cerr << "AC_ny=" << info.int_params[AC_ny] << "\n";
    std::cerr << "AC_nz=" << info.int_params[AC_nz] << "\n";
    std::cerr << "AC_mx=" << info.int_params[AC_mx] << "\n";
    std::cerr << "AC_my=" << info.int_params[AC_my] << "\n";
    std::cerr << "AC_mz=" << info.int_params[AC_mz] << "\n";
    std::cerr << "AC_nx_min=" << info.int_params[AC_nx_min] << "\n";
    std::cerr << "AC_ny_min=" << info.int_params[AC_ny_min] << "\n";
    std::cerr << "AC_nz_min=" << info.int_params[AC_nz_min] << "\n";
    std::cerr << "AC_nx_max=" << info.int_params[AC_nx_max] << "\n";
    std::cerr << "AC_ny_max=" << info.int_params[AC_ny_max] << "\n";
    std::cerr << "AC_nz_max=" << info.int_params[AC_nz_max] << "\n";
  }
  const int x = info.int_params[AC_nx];
  const int y = info.int_params[AC_ny];
  const int z = info.int_params[AC_nz];
  MPI_Barrier(MPI_COMM_WORLD);

  Method methods = Method::None;
  methods |= Method::CudaMpi;
  methods |= Method::ColoPackMemcpyUnpack;
  methods |= Method::CudaMemcpyPeer;
  methods |= Method::CudaKernel;
  if (Method::None == methods) {
    methods = Method::Default;
  }

  PlacementStrategy strategy = PlacementStrategy::NodeAware;
  if (trivialPlacement) {
    strategy = PlacementStrategy::Trivial;
  }

  Statistics iterTime, exchTime;

  { // scope domains before mpi_finalize
    size_t radius = 3;

    DistributedDomain dd(x, y, z);

    dd.set_methods(methods);
    dd.set_radius(radius);
    dd.set_placement(strategy);

    // add required data
    std::vector<DataHandle<AcReal>> handles;
    for (size_t i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
      handles.push_back(dd.add_data<AcReal>(""));
    }

    // create arrays
    std::cerr << "realize\n";
    dd.realize();

    MPI_Barrier(MPI_COMM_WORLD);

    // one stream for the interior, plus one stream for each exterior
    std::vector<RcStream> cStreamInterior(dd.domains().size());
    std::vector<std::vector<RcStream>> cStreamExterior(dd.domains().size());
    for (size_t di = 0; di < dd.domains().size(); ++di) {
      int device = dd.domains()[di].gpu();
      cStreamInterior[di] = RcStream(device);
      for (int i = 0; i < 26; ++i) { // 26 possible nbrs
        cStreamExterior[di].push_back(RcStream(device));
      }
    }

    // create mesh info for each device
    for (size_t di = 0; di < dd.domains().size(); ++di) {
      int device = dd.domains()[di].gpu();
      acDeviceLoadDefaultUniforms(device);

      std::cerr << info.int_params[AC_nx] << "\n";
      acDeviceLoadMeshInfo(device, info);
    }

    // create the VBAs for each domain
    std::vector<VertexBufferArray> vbas(dd.domains().size());
    for (size_t di = 0; di < dd.domains().size(); ++di) {
      VertexBufferArray &vba = vbas[di];
      LocalDomain &d = dd.domains()[di];

      for (size_t i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
        vba.in[i] = d.get_curr_accessor<AcReal>(handles[i]).ptr().ptr;
        vba.out[i] = d.get_next_accessor<AcReal>(handles[i]).ptr().ptr;
      }
    }

#if 0
    std::cerr << "init\n";
    for (size_t di = 0; di < dd.domains().size(); ++di) {
      auto &d = dd.domains()[di];
      d.set_device();
      dim3 dimBlock = Dim3::make_block_dim(d.raw_size(), 512);
      dim3 dimGrid = ((d.raw_size()) + Dim3(dimBlock) - 1) / (Dim3(dimBlock));
      init_kernel<<<dimGrid, dimBlock, 0, cStreamInterior[di]>>>(d.get_curr_accessor(dh0), d.get_compute_region(), 10);
      CUDA_RUNTIME(hipDeviceSynchronize());
    }

    if (0)
      dd.write_paraview("init");
#endif

    const std::vector<Rect3> interiors = dd.get_interior();
    const std::vector<std::vector<Rect3>> exteriors = dd.get_exterior();

    // stencil defines compute region in terms of grid points
    // while asteroth does it in terms of memory offset.
    // we will need to add in the offset from the stencil region
    const Dim3 acOff = Dim3(STENCIL_ORDER / 2, STENCIL_ORDER / 2, STENCIL_ORDER / 2);

    for (size_t iter = 0; iter < 5; ++iter) {

      double iterStart = MPI_Wtime();
      double exchElapsed = 0;

      for (int substep = 0; substep < 3; ++substep) {
        // launch operations on interior
        for (size_t di = 0; di < dd.domains().size(); ++di) {
          auto &d = dd.domains()[di];
          nvtxRangePush("launch");
          Rect3 cr = interiors[di];
          cr.lo += acOff - dd.get_origin(di); // astaroth indexing is memory offset based
          cr.hi += acOff - dd.get_origin(di);
          // std::cerr << rank << ": launch on region=" << cr << " (interior)\n";
          // std::cerr << src0.origin() << "=src0 origin\n";
          d.set_device();
          acDeviceLoadScalarUniform(d.gpu(), cStreamInterior[di], AC_dt, AC_REAL_EPSILON);
          integrate_substep(substep, cStreamInterior[di], cr, vbas[di]);
          nvtxRangePop(); // launch
        }

        // exchange halo
        std::cerr << rank << ": exchange\n";
        double exchStart = MPI_Wtime();
        dd.exchange();
        exchElapsed += MPI_Wtime() - exchStart;

        // launch on exteriors
        for (size_t di = 0; di < dd.domains().size(); ++di) {
          auto &d = dd.domains()[di];
          for (size_t si = 0; si < exteriors[di].size(); ++si) {
            nvtxRangePush("launch");
            Rect3 cr = exteriors[di][si];
            cr.lo += acOff - dd.get_origin(di); // astaroth indexing is memory offset based
            cr.hi += acOff - dd.get_origin(di);
            // std::cerr << rank << ": launch on region=" << cr << " (exterior)\n";
            // std::cerr << src0.origin() << "=src0 origin\n";
            d.set_device();
            integrate_substep(substep, cStreamExterior[di][si], cr, vbas[di]);
            nvtxRangePop(); // launch
            // CUDA_RUNTIME(hipDeviceSynchronize());
          }
        }

        // wait for stencil to complete
        for (auto &s : cStreamInterior) {
          CUDA_RUNTIME(hipStreamSynchronize(s));
        }
        for (auto &v : cStreamExterior) {
          for (auto &s : v) {
            CUDA_RUNTIME(hipStreamSynchronize(s));
          }
        }

        // swap inputs and outputs
        dd.swap();
      }

      double iterElapsed = MPI_Wtime() - iterStart;

      MPI_Allreduce(MPI_IN_PLACE, &iterElapsed, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
      MPI_Allreduce(MPI_IN_PLACE, &exchElapsed, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
      iterTime.insert(iterElapsed);
      exchTime.insert(exchElapsed);
    }

    if (0)
      dd.write_paraview("final");

  } // send domains out of scope before MPI_Finalize

  if (0 == rank) {
    std::cout << size;
    std::cout << "," << info.int_params[AC_nx];
    std::cout << "," << info.int_params[AC_ny];
    std::cout << "," << info.int_params[AC_nz];
    std::cout << "," << iterTime.trimean();
    std::cout << "," << exchTime.trimean();
    std::cout << "\n";
  }

  MPI_Finalize();

  return 0;
}
