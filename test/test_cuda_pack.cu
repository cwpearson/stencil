#include "catch2/catch.hpp"

// #include "stencil/copy.cuh"
#include "stencil/cuda_runtime.hpp"
#include "stencil/pack_kernel.cuh"
#include "stencil/pitched_ptr.hpp"

TEMPLATE_TEST_CASE("pack", "[pack][template]", int, double) {
  std::cerr << "TEST: \"pack -*\"\n";

  Dim3 arrSz(3, 4, 5);

  // src->dst and src -> dst -> dst2
  PitchedPtr<TestType> src;
  PitchedPtr<TestType> dst2;

  // 3*4*5 array
  CUDA_RUNTIME(hipSetDevice(0));
  INFO("alloc src");
  CUDA_RUNTIME(hipMallocManaged(&src.ptr, sizeof(TestType) * arrSz.x * arrSz.y * arrSz.z));
  INFO("alloc dst2");
  CUDA_RUNTIME(hipMallocManaged(&dst2.ptr, sizeof(TestType) * arrSz.x * arrSz.y * arrSz.z));
  src.xsize = arrSz.x * sizeof(TestType);
  src.pitch = arrSz.x * sizeof(TestType);
  src.ysize = arrSz.y;
  dst2.xsize = arrSz.x * sizeof(TestType);
  dst2.pitch = arrSz.x * sizeof(TestType);
  dst2.ysize = arrSz.y;

  /*
    z faces
    (z = 0)   (z = 1)   (z = 4)
    x ->      x ->      x ->
  y 0  1  2   12 13 14  48 49 50
  | 3  4  5   15 16 17  51 52 53
  v 6  7  8   18 19 20  54 55 56
    9 10 11   21 22 23  57 58 59

    x faces
    (x = 0)
    y ->
  z  0  3  6  9
  | 12 15 18 21
  v 24 27 30 33
    36 39 42 45
    48 51 54 57

    y faces
    (y = 0)       (y = 1)
    x ->          x ->
  z  0  1  2    z  3  4  5
  | 12 13 14    | 15 16 17
  v 24 25 26    v 27 28 29
    36 37 38      39 40 41
    48 49 50      51 52 53


  */

  INFO("set src");
  for (size_t zi = 0; zi < arrSz.z; ++zi) {
    for (size_t yi = 0; yi < arrSz.y; ++yi) {
      for (size_t xi = 0; xi < arrSz.x; ++xi) {
        src.ptr[zi * arrSz.y * arrSz.x + yi * arrSz.x + xi] = zi * arrSz.y * arrSz.x + yi * arrSz.x + xi;
      }
    }
  }
  INFO("dev sync");
  CUDA_RUNTIME(hipDeviceSynchronize());

  SECTION("pack z = 4") {
    std::cerr << "pack z = 4\n";
    TestType *dst = nullptr;
    CUDA_RUNTIME(hipMallocManaged(&dst, sizeof(TestType) * arrSz.x * arrSz.y));
    dim3 dimGrid(2,2,2);
    dim3 dimBlock(2,2,2);
    std::cerr << dimGrid << " " << dimBlock << "\n";
    pack_kernel<<<dimGrid, dimBlock>>>(dst, hipPitchedPtr(src), Dim3(0, 0, arrSz.z - 1), Dim3(arrSz.x, arrSz.y, 1),
                                       sizeof(TestType));
    CUDA_RUNTIME(hipDeviceSynchronize());

    REQUIRE(dst[0] == 48);
    REQUIRE(dst[1] == 49);
    REQUIRE(dst[11] == 59);

    SECTION("unpack") {
      CUDA_RUNTIME(hipMemset(dst2.ptr, 0, sizeof(TestType) * arrSz.x * arrSz.y * arrSz.z));
      unpack_kernel<<<dimGrid, dimBlock>>>(hipPitchedPtr(dst2), dst, Dim3(0, 0, arrSz.z - 1),
                                           Dim3(arrSz.x, arrSz.y, 1), sizeof(TestType));
      CUDA_RUNTIME(hipDeviceSynchronize());
      REQUIRE(dst2.ptr[48] == 48);
      REQUIRE(dst2.ptr[59] == 59);
    }

    CUDA_RUNTIME(hipFree(dst));
  }

  SECTION("pack x = 0") {
    TestType *dst = nullptr;
    CUDA_RUNTIME(hipMallocManaged(&dst, sizeof(TestType) * arrSz.y * arrSz.z));
    dim3 dimGrid(2, 2, 2);
    dim3 dimBlock(2, 2, 2);
    pack_kernel<<<dimGrid, dimBlock>>>(dst, hipPitchedPtr(src), Dim3(0, 0, 0), Dim3(1, arrSz.y, arrSz.z),
                                       sizeof(TestType));
    CUDA_RUNTIME(hipDeviceSynchronize());

    REQUIRE(dst[0] == 0);
    REQUIRE(dst[1] == 3);
    REQUIRE(dst[11] == 33);
    REQUIRE(dst[19] == 57);
    CUDA_RUNTIME(hipFree(dst));
  }

  SECTION("pack y = 1") {
    TestType *dst = nullptr;
    CUDA_RUNTIME(hipMallocManaged(&dst, sizeof(TestType) * arrSz.y * arrSz.z));
    dim3 dimGrid(2, 2, 2);
    dim3 dimBlock(2, 2, 2);
    pack_kernel<<<dimGrid, dimBlock>>>(dst, hipPitchedPtr(src), Dim3(0, 1, 0), Dim3(arrSz.x, 1, arrSz.z),
                                       sizeof(TestType));
    CUDA_RUNTIME(hipDeviceSynchronize());

    REQUIRE(dst[0] == 3);
    REQUIRE(dst[1] == 4);
    REQUIRE(dst[11] == 41);
    REQUIRE(dst[14] == 53);
    CUDA_RUNTIME(hipFree(dst));
  }

  CUDA_RUNTIME(hipFree(src.ptr));
  CUDA_RUNTIME(hipFree(dst2.ptr));
  CUDA_RUNTIME(hipDeviceSynchronize());
}

TEST_CASE("real cases", "[cuda]") {
  SECTION("30x40x50, radius 4, +x face") {
    size_t radius = 4;
    Dim3 arrSz(30, 40, 50);
    Dim3 rawSz(38, 48, 58);
    size_t elemSize = 4;

    char *buf;
    hipPitchedPtr dst;

    CUDA_RUNTIME(hipSetDevice(0));
    CUDA_RUNTIME(hipMallocManaged(&dst.ptr, elemSize * rawSz.x * rawSz.y * rawSz.z));
    dst.pitch = elemSize * rawSz.x;
    dst.xsize = elemSize * rawSz.x;
    dst.ysize = rawSz.y;
    CUDA_RUNTIME(hipMallocManaged(&buf, elemSize * radius * arrSz.y * arrSz.z));

    dim3 dimGrid(20, 20, 20);
    dim3 dimBlock(32, 4, 4);

    Dim3 haloPos(34, 4, 4);
    Dim3 haloExtent(4, 40, 50);

    unpack_kernel<<<dimGrid, dimBlock>>>(dst, buf, haloPos, haloExtent, elemSize);
    CUDA_RUNTIME(hipDeviceSynchronize());
    CUDA_RUNTIME(hipFree(buf));
    CUDA_RUNTIME(hipFree(dst.ptr));
  }
}
