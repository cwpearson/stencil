#include "catch2/catch.hpp"

#include <mpi.h>

#include "stencil/cuda_runtime.hpp"
#include "stencil/tx.hpp"

TEMPLATE_TEST_CASE("any tx", "[mpi][cuda][template]", int32_t, int64_t) {

  const size_t n = 100;

  TestType *buf0 = nullptr;
  TestType *buf1 = nullptr;

  INFO("allocate bufs");
  CUDA_RUNTIME(hipMallocManaged(&buf0, n * sizeof(TestType)));
  CUDA_RUNTIME(hipMallocManaged(&buf1, n * sizeof(TestType)));

  INFO("init bufs");
  for (size_t i = 0; i < n; ++i) {
    buf0[i] = i + 1;
    buf1[i] = 0;
  }
  REQUIRE(buf1[0] != buf0[0]);
  REQUIRE(buf1[n - 1] != buf0[n - 1]);

  INFO("ctors");
  int srcRank = 0;
  int srcGPU = 0;
  int dstRank = 0;
  int dstGPU = 0;
  size_t dataIdx = 0;
  AnySender sender(srcRank, srcGPU, dstRank, dstGPU, dataIdx, Dim3(0,0,0));
  AnyRecver recver(srcRank, srcGPU, dstRank, dstGPU, dataIdx, Dim3(0,0,0));

  
  INFO("resize");
  sender.resize(n * sizeof(TestType));
  recver.resize(n * sizeof(TestType));

  INFO("send/recv");
  sender.send(buf0);
  recver.recv(buf1);

  INFO("wait");
  sender.wait();
  recver.wait();

  INFO("cuda sync");
  CUDA_RUNTIME(hipDeviceSynchronize());

  REQUIRE(buf1[0] == buf0[0]);
  REQUIRE(buf1[n - 1] == buf0[n - 1]);

  CUDA_RUNTIME(hipFree(buf0));
  CUDA_RUNTIME(hipFree(buf1));
}