#include "hip/hip_runtime.h"
#include "catch2/catch.hpp"

#include <cstring> // std::memcpy

#include "stencil/copy.cuh"
#include "stencil/cuda_runtime.hpp"
#include "stencil/dim3.hpp"
#include "stencil/stencil.hpp"

__device__ int pack_xyz(int x, int y, int z) {
  int ret = 0;
  ret |= x & 0x3FF;
  ret |= (y & 0x3FF) << 10;
  ret |= (z & 0x3FF) << 20;
  return ret;
}

int unpack_x(int a) { return a & 0x3FF; }

int unpack_y(int a) { return (a >> 10) & 0x3FF; }

int unpack_z(int a) { return (a >> 20) & 0x3FF; }

template <typename T>
__global__ void
init_kernel(T *dst,          //<! [out] pointer to beginning of dst allocation
            const Dim3 rawSz //<! [in] 3D size of the dst and src allocations
) {

  constexpr size_t radius = 1;
  const Dim3 domSz = rawSz - Dim3(2 * radius, 2 * radius, 2 * radius);

  const size_t gdz = gridDim.z;
  const size_t biz = blockIdx.z;
  const size_t bdz = blockDim.z;
  const size_t tiz = threadIdx.z;

  const size_t gdy = gridDim.y;
  const size_t biy = blockIdx.y;
  const size_t bdy = blockDim.y;
  const size_t tiy = threadIdx.y;

  const size_t gdx = gridDim.x;
  const size_t bix = blockIdx.x;
  const size_t bdx = blockDim.x;
  const size_t tix = threadIdx.x;

#define _at(arr, _x, _y, _z) arr[_z * rawSz.y * rawSz.x + _y * rawSz.x + _x]

  // initialize the compute domain and set halos to zero
  for (size_t z = biz * bdz + tiz; z < rawSz.z; z += gdz * bdz) {
    for (size_t y = biy * bdy + tiy; y < rawSz.y; y += gdy * bdy) {
      for (size_t x = bix * bdx + tix; x < rawSz.x; x += gdx * bdx) {

        if (z >= radius && x >= radius && y >= radius && z < rawSz.z - radius &&
            y < rawSz.y - radius && x < rawSz.x - radius) {
          _at(dst, x, y, z) = pack_xyz(x, y, z);
        } else {
          _at(dst, x, y, z) = 0.0;
        }
      }
    }
  }

#undef _at
}

TEST_CASE("exchange") {

  size_t radius = 1;
  typedef float TestType1;

  INFO("ctor");
  DistributedDomain dd(10, 10, 10);
  dd.set_radius(radius);
  auto dh1 = dd.add_data<TestType1>();
  dd.set_methods(MethodFlags::CudaMpi);

  INFO("realize");
  dd.realize();

  CUDA_RUNTIME(hipDeviceSynchronize());

  INFO("barrier");
  MPI_Barrier(MPI_COMM_WORLD);

  INFO("init");
  dim3 dimGrid(10, 10, 10);
  dim3 dimBlock(8, 8, 8);
  for (auto &d : dd.domains()) {
    REQUIRE(d.get_curr(dh1) != nullptr);
    std::cerr << d.raw_size() << "\n";
    CUDA_RUNTIME(hipSetDevice(d.gpu()));
    init_kernel<<<dimGrid, dimBlock>>>(d.get_curr(dh1), d.raw_size());
    CUDA_RUNTIME(hipDeviceSynchronize());
  }

  MPI_Barrier(MPI_COMM_WORLD);

  // test initialization
  INFO("test init");
  for (auto &d : dd.domains()) {
    const Dim3 ext = d.halo_extent(Dim3(0, 0, 0));

    for (size_t qi = 0; qi < d.num_data(); ++qi) {
      auto vec = d.interior_to_host(qi);

      // make sure we can access data as a TestType1
      std::vector<TestType1> interior(ext.flatten());
      REQUIRE(vec.size() == interior.size() * sizeof(TestType1));
      std::memcpy(interior.data(), vec.data(), vec.size());

      for (int64_t z = 0; z < ext.z; ++z) {
        for (int64_t y = 0; y < ext.y; ++y) {
          for (int64_t x = 0; x < ext.x; ++x) {
            TestType1 val = interior[z * (ext.y * ext.x) + y * (ext.x) + x];
            REQUIRE(unpack_x(val) == x + radius);
            REQUIRE(unpack_y(val) == y + radius);
            REQUIRE(unpack_z(val) == z + radius);
          }
        }
      }
    }
  }

  MPI_Barrier(MPI_COMM_WORLD);

  INFO("exchange");
  dd.exchange();
  CUDA_RUNTIME(hipDeviceSynchronize());

  INFO("interior should be unchanged");
  for (auto &d : dd.domains()) {
    const Dim3 ext = d.halo_extent(Dim3(0, 0, 0));

    for (size_t qi = 0; qi < d.num_data(); ++qi) {
      auto vec = d.interior_to_host(qi);

      // make sure we can access data as a TestType1
      std::vector<TestType1> interior(ext.flatten());
      REQUIRE(vec.size() == interior.size() * sizeof(TestType1));
      std::memcpy(interior.data(), vec.data(), vec.size());

      for (int64_t z = 0; z < ext.z; ++z) {
        for (int64_t y = 0; y < ext.y; ++y) {
          for (int64_t x = 0; x < ext.x; ++x) {
            TestType1 val = interior[z * (ext.y * ext.x) + y * (ext.x) + x];
            REQUIRE(unpack_x(val) == x + radius);
            REQUIRE(unpack_y(val) == y + radius);
            REQUIRE(unpack_z(val) == z + radius);
          }
        }
      }
    }
  }
}
