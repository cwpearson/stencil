#include "hip/hip_runtime.h"
#include "catch2/catch.hpp"

#include <cstring> // std::memcpy

#include "stencil/accessor.hpp"
#include "stencil/copy.cuh"
#include "stencil/cuda_runtime.hpp"
#include "stencil/dim3.hpp"
#include "stencil/rect3.hpp"
#include "stencil/stencil.hpp"

template <typename T>
__global__ void init_kernel(Accessor<T> dst, //<! [out] region to fill
                            Rect3 dstExt     //<! [in] the extent of the region to initialize
) {
  const T ripple[4] = {0, 0.25, 0, -0.25};
  const size_t period = sizeof(ripple) / sizeof(ripple[0]);

  const size_t tiz = blockDim.z * blockIdx.z + threadIdx.z;
  const size_t tiy = blockDim.y * blockIdx.y + threadIdx.y;
  const size_t tix = blockDim.x * blockIdx.x + threadIdx.x;

  for (size_t z = dstExt.lo.z + tiz; z < dstExt.hi.z; z += gridDim.z * blockDim.z) {
    for (size_t y = dstExt.lo.y + tiy; y < dstExt.hi.y; y += gridDim.y * blockDim.y) {
      for (size_t x = dstExt.lo.x + tix; x < dstExt.hi.x; x += gridDim.x * blockDim.x) {

        Dim3 p(x, y, z);
        T val = p.x + ripple[p.x % period] + p.y + ripple[p.y % period] + p.z + ripple[p.z % period];
        dst[p] = val;
      }
    }
  }
}

/* check an exchange that supports the given kernel radius
 */
static void check_exchange(const Radius &radius) {

  int rank;
  int size;
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  MPI_Comm_size(MPI_COMM_WORLD, &size);

  srand(time(NULL) + rank);

  typedef float Q1;

  INFO("ctor");
  DistributedDomain dd(10, 10, 10);

  dd.set_radius(radius);
  auto dh1 = dd.add_data<Q1>("d0");
  dd.set_methods(MethodFlags::CudaMpi);

  INFO("realize");
  dd.realize();

  INFO("device sync");
  for (auto &d : dd.domains()) {
    CUDA_RUNTIME(hipSetDevice(d.gpu()));
    CUDA_RUNTIME(hipDeviceSynchronize());
  }

  INFO("barrier");
  MPI_Barrier(MPI_COMM_WORLD);

  INFO("init");
  dim3 dimGrid(10, 10, 10);
  dim3 dimBlock(8, 8, 8);
  for (auto &d : dd.domains()) {
    REQUIRE(d.get_curr(dh1) != nullptr);
    CUDA_RUNTIME(hipSetDevice(d.gpu()));
    auto acc = d.get_curr_accessor(dh1);

    std::cerr << "origin" << acc.origin() << "\n";
    std::cerr << "pitch " << acc.pitch() << "\n";
    Rect3 ext = d.get_compute_region();
    std::cout << "compute region " << ext << "\n";

    init_kernel<<<dimGrid, dimBlock>>>(acc, ext);
    CUDA_RUNTIME(hipDeviceSynchronize());
  }

  MPI_Barrier(MPI_COMM_WORLD);

  // test initialization
  INFO("test init");
  for (auto &d : dd.domains()) {
    const Dim3 origin = d.origin();
    const Dim3 ext = d.halo_extent(Dim3(0, 0, 0));

    for (size_t qi = 0; qi < d.num_data(); ++qi) {
      auto vec = d.interior_to_host(qi);

      // make sure we can access data as a Q1
      std::vector<Q1> interior(ext.flatten());
      REQUIRE(vec.size() == interior.size() * sizeof(Q1));
      std::memcpy(interior.data(), vec.data(), vec.size());

      // create an accessor for the CPU data
      Accessor<Q1> acc(interior.data(), origin, ext);
      Rect3 rect = d.get_compute_region();

      for (int64_t z = rect.lo.z; z < rect.hi.z; ++z) {
        for (int64_t y = rect.lo.y; y < rect.hi.y; ++y) {
          for (int64_t x = rect.lo.x; x < rect.hi.x; ++x) {
            Dim3 p(x, y, z);
            const Q1 ripple[4] = {0, 0.25, 0, -0.25};
            const size_t period = sizeof(ripple) / sizeof(ripple[0]);
            Q1 val = acc[p];
            REQUIRE(val == p.x + ripple[p.x % period] + p.y + ripple[p.y % period] + p.z + ripple[p.z % period]);
          }
        }
      }
    }
  }

  MPI_Barrier(MPI_COMM_WORLD);

  INFO("exchange");
  dd.exchange();
  CUDA_RUNTIME(hipDeviceSynchronize());
  MPI_Barrier(MPI_COMM_WORLD);

  INFO("check whole region after exchange");
  for (auto &d : dd.domains()) {

    const Rect3 region = d.get_full_region();
    const Dim3 ext = region.hi - region.lo;

    for (size_t qi = 0; qi < d.num_data(); ++qi) {
      auto vec = d.quantity_to_host(qi);
      // access quantity data as a Q1
      std::vector<Q1> quantity(ext.flatten());
      REQUIRE(vec.size() == quantity.size() * sizeof(Q1));
      std::memcpy(quantity.data(), vec.data(), vec.size());

      // create an accessor for the CPU data
      // the subdomain origin does not include the halo, but the access origin
      // is the point for the 0th offset in data
      Dim3 origin = d.origin();
      origin.x -= radius.x(-1);
      origin.y -= radius.y(-1);
      origin.z -= radius.z(-1);
      Accessor<Q1> acc(quantity.data(), origin, ext);
      Rect3 rect = d.get_full_region();

      std::cerr << "full region: " << rect << "\n";
      std::cerr << "compute region: " << d.get_compute_region() << "\n";
      std::cerr << "acc.origin()=" << acc.origin() << "\n";

      for (int64_t z = rect.lo.z; z < rect.hi.z; ++z) {
        for (int64_t y = rect.lo.y; y < rect.hi.y; ++y) {
          for (int64_t x = rect.lo.x; x < rect.hi.x; ++x) {
            Dim3 p(x, y, z);
            const Q1 ripple[4] = {0, 0.25, 0, -0.25};
            const size_t period = sizeof(ripple) / sizeof(ripple[0]);
            Q1 val = acc[p];

            // std::cerr << p;

            // if p is on one of the outside shell, should have recieved a value
            // from the interior of the opposite side
            if (p.x < 0) {
              p.x += dd.size().x;
            }
            if (p.y < 0) {
              p.y += dd.size().y;
            }
            if (p.z < 0) {
              p.z += dd.size().z;
            }
            if (p.x >= dd.size().x) {
              p.x -= dd.size().x;
            }
            if (p.y >= dd.size().y) {
              p.y -= dd.size().y;
            }
            if (p.z >= dd.size().z) {
              p.z -= dd.size().z;
            }

            // std::cerr << "->" << p << "\n";

            REQUIRE(val == p.x + ripple[p.x % period] + p.y + ripple[p.y % period] + p.z + ripple[p.z % period]);
          }
        }
      }
    }
  }
}

TEST_CASE("exchange2") {

  SECTION("r=0") { check_exchange(Radius::constant(0)); }

  SECTION("r=1") { check_exchange(Radius::constant(1)); }

  SECTION("r=2") { check_exchange(Radius::constant(2)); }

  SECTION("+x=2") {
    Radius r = Radius::constant(0);
    r.dir(1, 0, 0) = 2;
    check_exchange(r);
  }

  SECTION("mx=1") { // -x doesnt work as a section on CLI
    Radius r = Radius::constant(0);
    r.dir(-1, 0, 0) = 1;
    check_exchange(r);
  }

  SECTION("+x=2, mx=1") { // -x doesnt work as a section on CLI
    Radius r = Radius::constant(0);
    r.dir(1, 0, 0) = 2;
    r.dir(-1, 0, 0) = 1;
    check_exchange(r);
  }
}
