#include "catch2/catch.hpp"

#include <mpi.h>

#include "stencil/cuda_runtime.hpp"
#include "stencil/tx_cuda.cuh"

TEST_CASE("cudaipc", "[mpi][cuda]") {

  int myRank;
  int worldSize;
  MPI_Comm_rank(MPI_COMM_WORLD, &myRank);
  MPI_Comm_size(MPI_COMM_WORLD, &worldSize);

  if (myRank == 0) {
    // create event
    hipEvent_t event;
    CUDA_RUNTIME(hipEventCreate(&event, hipEventDisableTiming | hipEventInterprocess));

    // create handle
    hipIpcEventHandle_t handle;
    CUDA_RUNTIME(hipIpcGetEventHandle(&handle, event));

    // send handle to rank 1
    MPI_Send(&handle, sizeof(handle), MPI_BYTE, 1, 0, MPI_COMM_WORLD);

    MPI_Barrier(MPI_COMM_WORLD);

  } else if (1 == myRank) {

    // recv handle from rank 0
    hipIpcEventHandle_t handle;
    MPI_Recv(&handle, sizeof(handle), MPI_BYTE, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);

    // Create event from handle
    hipEvent_t event;
    CUDA_RUNTIME(hipIpcOpenEventHandle(&event, handle));

    MPI_Barrier(MPI_COMM_WORLD);

    CUDA_RUNTIME(hipEventDestroy(event));
  } else {
    MPI_Barrier(MPI_COMM_WORLD);
  }
  
}